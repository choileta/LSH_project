#include "hip/hip_runtime.h"
#include "lsh256.cuh"

__device__ void lsh256_core(uint32_t* state, uint32_t* hash) {

	//LOAD_MSG_BLK
	asm(".reg.b32               e_l<8>;");
	asm(".reg.b32               e_r<8>;");
	asm(".reg.b32               o_l<8>;");
	asm(".reg.b32               o_r<8>;");

	asm(".reg.b32				cv_l<8>;");
	asm(".reg.b32				cv_r<8>;");
	asm(".reg.b32				tmp<1>;");

	asm("mov.b32 	            cv_l0,        %0;"		:: "r"(hash[0]));
	asm("mov.b32 	            cv_l1,        %0;"		:: "r"(hash[1]));
	asm("mov.b32 	            cv_l2,        %0;"		:: "r"(hash[2]));
	asm("mov.b32 	            cv_l3,        %0;"		:: "r"(hash[3]));
	asm("mov.b32 	            cv_l4,        %0;"		:: "r"(hash[4]));
	asm("mov.b32 	            cv_l5,        %0;"		:: "r"(hash[5]));
	asm("mov.b32 	            cv_l6,        %0;"		:: "r"(hash[6]));
	asm("mov.b32 	            cv_l7,        %0;"		:: "r"(hash[7]));

	asm("mov.b32 	            cv_r0,        %0;"		:: "r"(hash[8]));
	asm("mov.b32 	            cv_r1,        %0;"		:: "r"(hash[9]));
	asm("mov.b32 	            cv_r2,        %0;"		:: "r"(hash[10]));
	asm("mov.b32 	            cv_r3,        %0;"		:: "r"(hash[11]));
	asm("mov.b32 	            cv_r4,        %0;"		:: "r"(hash[12]));
	asm("mov.b32 	            cv_r5,        %0;"		:: "r"(hash[13]));
	asm("mov.b32 	            cv_r6,        %0;"		:: "r"(hash[14]));
	asm("mov.b32 	            cv_r7,        %0;"		:: "r"(hash[15]));

	asm("mov.b32 	            e_l0,        %0;"		:: "r"(state[0]));
	asm("mov.b32 	            e_l1,        %0;"		:: "r"(state[1]));
	asm("mov.b32 	            e_l2,        %0;"		:: "r"(state[2]));
	asm("mov.b32 	            e_l3,        %0;"		:: "r"(state[3]));
	asm("mov.b32 	            e_l4,        %0;"		:: "r"(state[4]));
	asm("mov.b32 	            e_l5,        %0;"		:: "r"(state[5]));
	asm("mov.b32 	            e_l6,        %0;"		:: "r"(state[6]));
	asm("mov.b32 	            e_l7,        %0;"		:: "r"(state[7]));

	asm("mov.b32 	            e_r0,       %0;"		:: "r"(state[8]));
	asm("mov.b32 	            e_r1,       %0;"		:: "r"(state[9]));
	asm("mov.b32 	            e_r2,       %0;"		:: "r"(state[10]));
	asm("mov.b32 	            e_r3,       %0;"		:: "r"(state[11]));
	asm("mov.b32 	            e_r4,       %0;"		:: "r"(state[12]));
	asm("mov.b32 	            e_r5,       %0;"		:: "r"(state[13]));
	asm("mov.b32 	            e_r6,       %0;"		:: "r"(state[14]));
	asm("mov.b32 	            e_r7,       %0;"		:: "r"(state[15]));


	asm("mov.b32 	            o_l0,       %0;"		:: "r"(state[16]));
	asm("mov.b32 	            o_l1,       %0;"		:: "r"(state[17]));
	asm("mov.b32 	            o_l2,       %0;"		:: "r"(state[18]));
	asm("mov.b32 	            o_l3,       %0;"		:: "r"(state[19]));
	asm("mov.b32 	            o_l4,       %0;"		:: "r"(state[20]));
	asm("mov.b32 	            o_l5,       %0;"		:: "r"(state[21]));
	asm("mov.b32 	            o_l6,       %0;"		:: "r"(state[22]));
	asm("mov.b32 	            o_l7,       %0;"		:: "r"(state[23]));

	asm("mov.b32 	            o_r0,       %0;"		:: "r"(state[24]));
	asm("mov.b32 	            o_r1,       %0;"		:: "r"(state[25]));
	asm("mov.b32 	            o_r2,       %0;"		:: "r"(state[26]));
	asm("mov.b32 	            o_r3,       %0;"		:: "r"(state[27]));
	asm("mov.b32 	            o_r4,       %0;"		:: "r"(state[28]));
	asm("mov.b32 	            o_r5,       %0;"		:: "r"(state[29]));
	asm("mov.b32 	            o_r6,       %0;"		:: "r"(state[30]));
	asm("mov.b32 	            o_r7,       %0;"		:: "r"(state[31]));

	//! MSG_ADD_EVEN
	//cv_l part
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");
	//!MIX(0)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x917caf90;");
	asm("xor.b32				cv_l1, cv_l1, 0x6c1b10a2;");
	asm("xor.b32				cv_l2, cv_l2, 0x6f352943;");
	asm("xor.b32				cv_l3, cv_l3, 0xcf778243;");
	asm("xor.b32				cv_l4, cv_l4, 0x2ceb7472;");
	asm("xor.b32				cv_l5, cv_l5, 0x29e96ff2;");
	asm("xor.b32				cv_l6, cv_l6, 0x8a9ba428;");
	asm("xor.b32				cv_l7, cv_l7, 0x2eeb2642;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(1)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x0e2c4021;");
	asm("xor.b32				cv_l1, cv_l1, 0x872bb30e;");
	asm("xor.b32				cv_l2, cv_l2, 0xa45e6cb2;");
	asm("xor.b32				cv_l3, cv_l3, 0x46f9c612;");
	asm("xor.b32				cv_l4, cv_l4, 0x185fe69e;");
	asm("xor.b32				cv_l5, cv_l5, 0x1359621b;");
	asm("xor.b32				cv_l6, cv_l6, 0x263fccb2;");
	asm("xor.b32				cv_l7, cv_l7, 0x1a116870;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//loop (1)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x3a6c612f;");
	asm("xor.b32				cv_l1, cv_l1, 0xb2dec195;");
	asm("xor.b32				cv_l2, cv_l2, 0x02cb1f56;");
	asm("xor.b32				cv_l3, cv_l3, 0x40bfd858;");
	asm("xor.b32				cv_l4, cv_l4, 0x784684b6;");
	asm("xor.b32				cv_l5, cv_l5, 0x6cbb7d2e;");
	asm("xor.b32				cv_l6, cv_l6, 0x660c7ed8;");
	asm("xor.b32				cv_l7, cv_l7, 0x2b79d88a;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0xa6cd9069;");
	asm("xor.b32				cv_l1, cv_l1, 0x91a05747;");
	asm("xor.b32				cv_l2, cv_l2, 0xcdea7558;");
	asm("xor.b32				cv_l3, cv_l3, 0x00983098;");
	asm("xor.b32				cv_l4, cv_l4, 0xbecb3b2e;");
	asm("xor.b32				cv_l5, cv_l5, 0x2838ab9a;");
	asm("xor.b32				cv_l6, cv_l6, 0x728b573e;");
	asm("xor.b32				cv_l7, cv_l7, 0xa55262b5;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");


	//loop (2)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x745dfa0f;");
	asm("xor.b32				cv_l1, cv_l1, 0x31f79ed8;");
	asm("xor.b32				cv_l2, cv_l2, 0xb85fce25;");
	asm("xor.b32				cv_l3, cv_l3, 0x98c8c898;");
	asm("xor.b32				cv_l4, cv_l4, 0x8a0669ec;");
	asm("xor.b32				cv_l5, cv_l5, 0x60e445c2;");
	asm("xor.b32				cv_l6, cv_l6, 0xfde295b0;");
	asm("xor.b32				cv_l7, cv_l7, 0xf7b5185a;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0xd2580983;");
	asm("xor.b32				cv_l1, cv_l1, 0x29967709;");
	asm("xor.b32				cv_l2, cv_l2, 0x182df3dd;");
	asm("xor.b32				cv_l3, cv_l3, 0x61916130;");
	asm("xor.b32				cv_l4, cv_l4, 0x90705676;");
	asm("xor.b32				cv_l5, cv_l5, 0x452a0822;");
	asm("xor.b32				cv_l6, cv_l6, 0xe07846ad;");
	asm("xor.b32				cv_l7, cv_l7, 0xaccd7351;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");


	//loop (3)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x2a618d55;");
	asm("xor.b32				cv_l1, cv_l1, 0xc00d8032;");
	asm("xor.b32				cv_l2, cv_l2, 0x4621d0f5;");
	asm("xor.b32				cv_l3, cv_l3, 0xf2f29191;");
	asm("xor.b32				cv_l4, cv_l4, 0x00c6cd06;");
	asm("xor.b32				cv_l5, cv_l5, 0x6f322a67;");
	asm("xor.b32				cv_l6, cv_l6, 0x58bef48d;");
	asm("xor.b32				cv_l7, cv_l7, 0x7a40c4fd;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x8beee27f;");
	asm("xor.b32				cv_l1, cv_l1, 0xcd8db2f2;");
	asm("xor.b32				cv_l2, cv_l2, 0x67f2c63b;");
	asm("xor.b32				cv_l3, cv_l3, 0xe5842383;");
	asm("xor.b32				cv_l4, cv_l4, 0xc793d306;");
	asm("xor.b32				cv_l5, cv_l5, 0xa15c91d6;");
	asm("xor.b32				cv_l6, cv_l6, 0x17b381e5;");
	asm("xor.b32				cv_l7, cv_l7, 0xbb05c277;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");



	//loop (4)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x7ad1620a;");
	asm("xor.b32				cv_l1, cv_l1, 0x5b40a5bf;");
	asm("xor.b32				cv_l2, cv_l2, 0x5ab901a2;");
	asm("xor.b32				cv_l3, cv_l3, 0x69a7a768;");
	asm("xor.b32				cv_l4, cv_l4, 0x5b66d9cd;");
	asm("xor.b32				cv_l5, cv_l5, 0xfdee6877;");
	asm("xor.b32				cv_l6, cv_l6, 0xcb3566fc;");
	asm("xor.b32				cv_l7, cv_l7, 0xc0c83a32;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x4c336c84;");
	asm("xor.b32				cv_l1, cv_l1, 0x9be6651a;");
	asm("xor.b32				cv_l2, cv_l2, 0x13baa3fc;");
	asm("xor.b32				cv_l3, cv_l3, 0x114f0fd1;");
	asm("xor.b32				cv_l4, cv_l4, 0xc240a728;");
	asm("xor.b32				cv_l5, cv_l5, 0xec56e074;");
	asm("xor.b32				cv_l6, cv_l6, 0x009c63c7;");
	asm("xor.b32				cv_l7, cv_l7, 0x89026cf2;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");


	//loop (5)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x7f9ff0d0;");
	asm("xor.b32				cv_l1, cv_l1, 0x824b7fb5;");
	asm("xor.b32				cv_l2, cv_l2, 0xce5ea00f;");
	asm("xor.b32				cv_l3, cv_l3, 0x605ee0e2;");
	asm("xor.b32				cv_l4, cv_l4, 0x02e7cfea;");
	asm("xor.b32				cv_l5, cv_l5, 0x43375560;");
	asm("xor.b32				cv_l6, cv_l6, 0x9d002ac7;");
	asm("xor.b32				cv_l7, cv_l7, 0x8b6f5f7b;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x1f90c14f;");
	asm("xor.b32				cv_l1, cv_l1, 0xcdcb3537;");
	asm("xor.b32				cv_l2, cv_l2, 0x2cfeafdd;");
	asm("xor.b32				cv_l3, cv_l3, 0xbf3fc342;");
	asm("xor.b32				cv_l4, cv_l4, 0xeab7b9ec;");
	asm("xor.b32				cv_l5, cv_l5, 0x7a8cb5a3;");
	asm("xor.b32				cv_l6, cv_l6, 0x9d2af264;");
	asm("xor.b32				cv_l7, cv_l7, 0xfacedb06;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");


	//loop (6)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0xb052106e;");
	asm("xor.b32				cv_l1, cv_l1, 0x99006d04;");
	asm("xor.b32				cv_l2, cv_l2, 0x2bae8d09;");
	asm("xor.b32				cv_l3, cv_l3, 0xff030601;");
	asm("xor.b32				cv_l4, cv_l4, 0xa271a6d6;");
	asm("xor.b32				cv_l5, cv_l5, 0x0742591d;");
	asm("xor.b32				cv_l6, cv_l6, 0xc81d5701;");
	asm("xor.b32				cv_l7, cv_l7, 0xc9a9e200;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x02627f1e;");
	asm("xor.b32				cv_l1, cv_l1, 0x996d719d;");
	asm("xor.b32				cv_l2, cv_l2, 0xda3b9634;");
	asm("xor.b32				cv_l3, cv_l3, 0x02090800;");
	asm("xor.b32				cv_l4, cv_l4, 0x14187d78;");
	asm("xor.b32				cv_l5, cv_l5, 0x499b7624;");
	asm("xor.b32				cv_l6, cv_l6, 0xe57458c9;");
	asm("xor.b32				cv_l7, cv_l7, 0x738be2c9;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//loop (7)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x64e19d20;");
	asm("xor.b32				cv_l1, cv_l1, 0x06df0f36;");
	asm("xor.b32				cv_l2, cv_l2, 0x15d1cb0e;");
	asm("xor.b32				cv_l3, cv_l3, 0x0b110802;");
	asm("xor.b32				cv_l4, cv_l4, 0x2c95f58c;");
	asm("xor.b32				cv_l5, cv_l5, 0xe5119a6d;");
	asm("xor.b32				cv_l6, cv_l6, 0x59cd22ae;");
	asm("xor.b32				cv_l7, cv_l7, 0xff6eac3c;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x467ebd84;");
	asm("xor.b32				cv_l1, cv_l1, 0xe5ee453c;");
	asm("xor.b32				cv_l2, cv_l2, 0xe79cd923;");
	asm("xor.b32				cv_l3, cv_l3, 0x1c190a0d;");
	asm("xor.b32				cv_l4, cv_l4, 0xc28b81b8;");
	asm("xor.b32				cv_l5, cv_l5, 0xf6ac0852;");
	asm("xor.b32				cv_l6, cv_l6, 0x26efd107;");
	asm("xor.b32				cv_l7, cv_l7, 0x6e1ae93b;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");


	//loop (8)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0xc53c41ca;");
	asm("xor.b32				cv_l1, cv_l1, 0xd4338221;");
	asm("xor.b32				cv_l2, cv_l2, 0x8475fd0a;");
	asm("xor.b32				cv_l3, cv_l3, 0x35231729;");
	asm("xor.b32				cv_l4, cv_l4, 0x4e0d3a7a;");
	asm("xor.b32				cv_l5, cv_l5, 0xa2b45b48;");
	asm("xor.b32				cv_l6, cv_l6, 0x16c0d82d;");
	asm("xor.b32				cv_l7, cv_l7, 0x890424a9;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x017e0c8f;");
	asm("xor.b32				cv_l1, cv_l1, 0x07b5a3f5;");
	asm("xor.b32				cv_l2, cv_l2, 0xfa73078e;");
	asm("xor.b32				cv_l3, cv_l3, 0x583a405e;");
	asm("xor.b32				cv_l4, cv_l4, 0x5b47b4c8;");
	asm("xor.b32				cv_l5, cv_l5, 0x570fa3ea;");
	asm("xor.b32				cv_l6, cv_l6, 0xd7990543;");
	asm("xor.b32				cv_l7, cv_l7, 0x8d28ce32;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");


	//loop (9)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x7f8a9b90;");
	asm("xor.b32				cv_l1, cv_l1, 0xbd5998fc;");
	asm("xor.b32				cv_l2, cv_l2, 0x6d7a9688;");
	asm("xor.b32				cv_l3, cv_l3, 0x927a9eb6;");
	asm("xor.b32				cv_l4, cv_l4, 0xa2fc7d23;");
	asm("xor.b32				cv_l5, cv_l5, 0x66b38e41;");
	asm("xor.b32				cv_l6, cv_l6, 0x709e491a;");
	asm("xor.b32				cv_l7, cv_l7, 0xb5f700bf;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x0a262c0f;");
	asm("xor.b32				cv_l1, cv_l1, 0x16f295b9;");
	asm("xor.b32				cv_l2, cv_l2, 0xe8111ef5;");
	asm("xor.b32				cv_l3, cv_l3, 0x0d195548;");
	asm("xor.b32				cv_l4, cv_l4, 0x9f79a0c5;");
	asm("xor.b32				cv_l5, cv_l5, 0x1a41cfa7;");
	asm("xor.b32				cv_l6, cv_l6, 0x0ee7638a;");
	asm("xor.b32				cv_l7, cv_l7, 0xacf7c074;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//loop (10)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x30523b19;");
	asm("xor.b32				cv_l1, cv_l1, 0x09884ecf;");
	asm("xor.b32				cv_l2, cv_l2, 0xf93014dd;");
	asm("xor.b32				cv_l3, cv_l3, 0x266e9d55;");
	asm("xor.b32				cv_l4, cv_l4, 0x191a6664;");
	asm("xor.b32				cv_l5, cv_l5, 0x5c1176c1;");
	asm("xor.b32				cv_l6, cv_l6, 0xf64aed98;");
	asm("xor.b32				cv_l7, cv_l7, 0xa4b83520;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x828d5449;");
	asm("xor.b32				cv_l1, cv_l1, 0x91d71dd8;");
	asm("xor.b32				cv_l2, cv_l2, 0x2944f2d6;");
	asm("xor.b32				cv_l3, cv_l3, 0x950bf27b;");
	asm("xor.b32				cv_l4, cv_l4, 0x3380ca7d;");
	asm("xor.b32				cv_l5, cv_l5, 0x6d88381d;");
	asm("xor.b32				cv_l6, cv_l6, 0x4138868e;");
	asm("xor.b32				cv_l7, cv_l7, 0x5ced55c4;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//loop (11)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x0fe19dcb;");
	asm("xor.b32				cv_l1, cv_l1, 0x68f4f669;");
	asm("xor.b32				cv_l2, cv_l2, 0x6e37c8ff;");
	asm("xor.b32				cv_l3, cv_l3, 0xa0fe6e10;");
	asm("xor.b32				cv_l4, cv_l4, 0xb44b47b0;");
	asm("xor.b32				cv_l5, cv_l5, 0xf5c0558a;");
	asm("xor.b32				cv_l6, cv_l6, 0x79bf14cf;");
	asm("xor.b32				cv_l7, cv_l7, 0x4a431a20;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0xf17f68da;");
	asm("xor.b32				cv_l1, cv_l1, 0x5deb5fd1;");
	asm("xor.b32				cv_l2, cv_l2, 0xa600c86d;");
	asm("xor.b32				cv_l3, cv_l3, 0x9f6c7eb0;");
	asm("xor.b32				cv_l4, cv_l4, 0xff92f864;");
	asm("xor.b32				cv_l5, cv_l5, 0xb615e07f;");
	asm("xor.b32				cv_l6, cv_l6, 0x38d3e448;");
	asm("xor.b32				cv_l7, cv_l7, 0x8d5d3a6a;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//loop (12)
	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//! msg_add_even(i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//!MIX(2)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 29)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 29;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 29;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 29;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 29;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 29;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 29;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 29;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 29;");

	//xor_with_const
	asm("xor.b32				cv_l0, cv_l0, 0x70e843cb;");
	asm("xor.b32				cv_l1, cv_l1, 0x494b312e;");
	asm("xor.b32				cv_l2, cv_l2, 0xa6c93613;");
	asm("xor.b32				cv_l3, cv_l3, 0x0beb2f4f;");
	asm("xor.b32				cv_l4, cv_l4, 0x928b5d63;");
	asm("xor.b32				cv_l5, cv_l5, 0xcbf66035;");
	asm("xor.b32				cv_l6, cv_l6, 0x0cb82c80;");
	asm("xor.b32				cv_l7, cv_l7, 0xea97a4f7;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 1)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 1;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 1;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 1;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 1;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 1;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 1;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 1;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 1;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");

	//!msg_exp_odd(i_state)
	asm("mov.b32 	            tmp0, o_l0;");
	asm("add.u32				o_l0, e_l0, o_l3;");
	asm("add.u32				o_l3, e_l3, o_l1;");
	asm("add.u32				o_l1, e_l1, o_l2;");
	asm("add.u32				o_l2, e_l2, tmp0;");

	asm("mov.b32 	            tmp0, o_l4;");
	asm("add.u32				o_l4, e_l4, o_l7;");
	asm("add.u32				o_l7, e_l7, o_l6;");
	asm("add.u32				o_l6, e_l6, o_l5;");
	asm("add.u32				o_l5, e_l5, tmp0;");

	asm("mov.b32 	            tmp0, o_r0;");
	asm("add.u32				o_r0, e_r0, o_r3;");
	asm("add.u32				o_r3, e_r3, o_r1;");
	asm("add.u32				o_r1, e_r1, o_r2;");
	asm("add.u32				o_r2, e_r2, tmp0;");

	asm("mov.b32 	            tmp0, o_r4;");
	asm("add.u32				o_r4, e_r4, o_r7;");
	asm("add.u32				o_r7, e_r7, o_r6;");
	asm("add.u32				o_r6, e_r6, o_r5;");
	asm("add.u32				o_r5, e_r5, tmp0;");

	//!msg_add_odd(cv_l, cv_r, i_state)
	//cv_l part
	asm("xor.b32				cv_l0, o_l0, cv_l0;");
	asm("xor.b32				cv_l1, o_l1, cv_l1;");
	asm("xor.b32				cv_l2, o_l2, cv_l2;");
	asm("xor.b32				cv_l3, o_l3, cv_l3;");
	asm("xor.b32				cv_l4, o_l4, cv_l4;");
	asm("xor.b32				cv_l5, o_l5, cv_l5;");
	asm("xor.b32				cv_l6, o_l6, cv_l6;");
	asm("xor.b32				cv_l7, o_l7, cv_l7;");
	//cv_r part
	asm("xor.b32				cv_r0, o_r0, cv_r0;");
	asm("xor.b32				cv_r1, o_r1, cv_r1;");
	asm("xor.b32				cv_r2, o_r2, cv_r2;");
	asm("xor.b32				cv_r3, o_r3, cv_r3;");
	asm("xor.b32				cv_r4, o_r4, cv_r4;");
	asm("xor.b32				cv_r5, o_r5, cv_r5;");
	asm("xor.b32				cv_r6, o_r6, cv_r6;");
	asm("xor.b32				cv_r7, o_r7, cv_r7;");

	//!MIX(4)
	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_blk(cv_l, 5)
	asm("shf.l.wrap.b32         cv_l0, cv_l0, cv_l0, 5;");
	asm("shf.l.wrap.b32         cv_l1, cv_l1, cv_l1, 5;");
	asm("shf.l.wrap.b32         cv_l2, cv_l2, cv_l2, 5;");
	asm("shf.l.wrap.b32         cv_l3, cv_l3, cv_l3, 5;");
	asm("shf.l.wrap.b32         cv_l4, cv_l4, cv_l4, 5;");
	asm("shf.l.wrap.b32         cv_l5, cv_l5, cv_l5, 5;");
	asm("shf.l.wrap.b32         cv_l6, cv_l6, cv_l6, 5;");
	asm("shf.l.wrap.b32         cv_l7, cv_l7, cv_l7, 5;");

	//xor_with_const(cv_l, const_v)
	asm("xor.b32				cv_l0, cv_l0, 0x592c0f3b;");
	asm("xor.b32				cv_l1, cv_l1, 0x947c5f77;");
	asm("xor.b32				cv_l2, cv_l2, 0x6fff49b9;");
	asm("xor.b32				cv_l3, cv_l3, 0xf71a7e5a;");
	asm("xor.b32				cv_l4, cv_l4, 0x1de8c0f5;");
	asm("xor.b32				cv_l5, cv_l5, 0xc2569600;");
	asm("xor.b32				cv_l6, cv_l6, 0xc4e4ac8c;");
	asm("xor.b32				cv_l7, cv_l7, 0x823c9ce1;");

	//add_blk(cv_r, cv_l)
	asm("add.u32				cv_r0, cv_l0, cv_r0;");
	asm("add.u32				cv_r1, cv_l1, cv_r1;");
	asm("add.u32				cv_r2, cv_l2, cv_r2;");
	asm("add.u32				cv_r3, cv_l3, cv_r3;");
	asm("add.u32				cv_r4, cv_l4, cv_r4;");
	asm("add.u32				cv_r5, cv_l5, cv_r5;");
	asm("add.u32				cv_r6, cv_l6, cv_r6;");
	asm("add.u32				cv_r7, cv_l7, cv_r7;");

	//rotate_blk(cv_r, 17)
	asm("shf.l.wrap.b32         cv_r0, cv_r0, cv_r0, 17;");
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 17;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 17;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 17;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 17;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 17;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 17;");
	asm("shf.l.wrap.b32         cv_r7, cv_r7, cv_r7, 17;");

	//add_blk(cv_l, cv_r)
	asm("add.u32				cv_l0, cv_l0, cv_r0;");
	asm("add.u32				cv_l1, cv_l1, cv_r1;");
	asm("add.u32				cv_l2, cv_l2, cv_r2;");
	asm("add.u32				cv_l3, cv_l3, cv_r3;");
	asm("add.u32				cv_l4, cv_l4, cv_r4;");
	asm("add.u32				cv_l5, cv_l5, cv_r5;");
	asm("add.u32				cv_l6, cv_l6, cv_r6;");
	asm("add.u32				cv_l7, cv_l7, cv_r7;");

	//rotate_msg_gmma(cv_r)
	asm("shf.l.wrap.b32         cv_r1, cv_r1, cv_r1, 8;");
	asm("shf.l.wrap.b32         cv_r2, cv_r2, cv_r2, 16;");
	asm("shf.l.wrap.b32         cv_r3, cv_r3, cv_r3, 24;");
	asm("shf.l.wrap.b32         cv_r4, cv_r4, cv_r4, 24;");
	asm("shf.l.wrap.b32         cv_r5, cv_r5, cv_r5, 16;");
	asm("shf.l.wrap.b32         cv_r6, cv_r6, cv_r6, 8;");

	//!word_perm(cv_l, cv_r)
	asm("mov.b32 	            tmp0, cv_l0;");
	asm("mov.b32 	            cv_l0, cv_l6;");
	asm("mov.b32 	            cv_l6, cv_r6;");
	asm("mov.b32 	            cv_r6, cv_r2;");
	asm("mov.b32 	            cv_r2, cv_l1;");
	asm("mov.b32 	            cv_l1, cv_l4;");
	asm("mov.b32 	            cv_l4, cv_r4;");
	asm("mov.b32 	            cv_r4, cv_r0;");
	asm("mov.b32 	            cv_r0, cv_l2;");
	asm("mov.b32 	            cv_l2, cv_l5;");
	asm("mov.b32 	            cv_l5, cv_r7;");
	asm("mov.b32 	            cv_r7, cv_r1;");
	asm("mov.b32 	            cv_r1, tmp0;");
	asm("mov.b32 	            tmp0, cv_l3;");
	asm("mov.b32 	            cv_l3, cv_l7;");
	asm("mov.b32 	            cv_l7, cv_r5;");
	asm("mov.b32 	            cv_r5, cv_r3;");
	asm("mov.b32 	            cv_r3, tmp0;");
	//loop end

	//msg_exp_even(i_state)
	asm("mov.b32 	            tmp0, e_l0;");
	asm("add.u32				e_l0, o_l0, e_l3;");
	asm("add.u32				e_l3, o_l3, e_l1;");
	asm("add.u32				e_l1, o_l1, e_l2;");
	asm("add.u32				e_l2, o_l2, tmp0;");

	asm("mov.b32 	            tmp0, e_l4;");
	asm("add.u32				e_l4, o_l4, e_l7;");
	asm("add.u32				e_l7, o_l7, e_l6;");
	asm("add.u32				e_l6, o_l6, e_l5;");
	asm("add.u32				e_l5, o_l5, tmp0;");

	asm("mov.b32 	            tmp0, e_r0;");
	asm("add.u32				e_r0, o_r0, e_r3;");
	asm("add.u32				e_r3, o_r3, e_r1;");
	asm("add.u32				e_r1, o_r1, e_r2;");
	asm("add.u32				e_r2, o_r2, tmp0;");

	asm("mov.b32 	            tmp0, e_r4;");
	asm("add.u32				e_r4, o_r4, e_r7;");
	asm("add.u32				e_r7, o_r7, e_r6;");
	asm("add.u32				e_r6, o_r6, e_r5;");
	asm("add.u32				e_r5, o_r5, tmp0;");

	//msg_add_even(cv_l, cv_r, i_state)
	asm("xor.b32				cv_l0, e_l0, cv_l0;");
	asm("xor.b32				cv_l1, e_l1, cv_l1;");
	asm("xor.b32				cv_l2, e_l2, cv_l2;");
	asm("xor.b32				cv_l3, e_l3, cv_l3;");
	asm("xor.b32				cv_l4, e_l4, cv_l4;");
	asm("xor.b32				cv_l5, e_l5, cv_l5;");
	asm("xor.b32				cv_l6, e_l6, cv_l6;");
	asm("xor.b32				cv_l7, e_l7, cv_l7;");
	asm("xor.b32				cv_r0, e_r0, cv_r0;");
	asm("xor.b32				cv_r1, e_r1, cv_r1;");
	asm("xor.b32				cv_r2, e_r2, cv_r2;");
	asm("xor.b32				cv_r3, e_r3, cv_r3;");
	asm("xor.b32				cv_r4, e_r4, cv_r4;");
	asm("xor.b32				cv_r5, e_r5, cv_r5;");
	asm("xor.b32				cv_r6, e_r6, cv_r6;");
	asm("xor.b32				cv_r7, e_r7, cv_r7;");

	//data push
	asm("mov.b32 	            %0,  cv_l0;": "+r"(hash[0]));
	asm("mov.b32 	            %0,  cv_l1;": "+r"(hash[1]));
	asm("mov.b32 	            %0,  cv_l2;": "+r"(hash[2]));
	asm("mov.b32 	            %0,  cv_l3;": "+r"(hash[3]));
	asm("mov.b32 	            %0,  cv_l4;": "+r"(hash[4]));
	asm("mov.b32 	            %0,  cv_l5;": "+r"(hash[5]));
	asm("mov.b32 	            %0,  cv_l6;": "+r"(hash[6]));
	asm("mov.b32 	            %0,  cv_l7;": "+r"(hash[7]));

	asm("mov.b32 	            %0,  cv_r0;": "+r"(hash[8]));
	asm("mov.b32 	            %0,  cv_r1;": "+r"(hash[9]));
	asm("mov.b32 	            %0,  cv_r2;": "+r"(hash[10]));
	asm("mov.b32 	            %0,  cv_r3;": "+r"(hash[11]));
	asm("mov.b32 	            %0,  cv_r4;": "+r"(hash[12]));
	asm("mov.b32 	            %0,  cv_r5;": "+r"(hash[13]));
	asm("mov.b32 	            %0,  cv_r6;": "+r"(hash[14]));
	asm("mov.b32 	            %0,  cv_r7;": "+r"(hash[15]));

}
__device__ void lsh256_final_core(uint32_t* hash_val) {
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[0]) : "r"(hash_val[0]), "r"(hash_val[8]));
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[1]) : "r"(hash_val[1]), "r"(hash_val[9]));
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[2]) : "r"(hash_val[2]), "r"(hash_val[10]));
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[3]) : "r"(hash_val[3]), "r"(hash_val[11]));
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[4]) : "r"(hash_val[4]), "r"(hash_val[12]));
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[5]) : "r"(hash_val[5]), "r"(hash_val[13]));
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[6]) : "r"(hash_val[6]), "r"(hash_val[14]));
	asm("xor.b32				%0, %1, %2;": "+r"(hash_val[7]) : "r"(hash_val[7]), "r"(hash_val[15]));
}
__device__ void lsh256_init(LSH256_info* info) {
	for (int i = 0; i < LSH_BLOCK_BYTE_LEN; i++)
		info->sv_pt[i] = 0;
	info->remain_byte_len = 0;

	info->hash_val[0] = 0x46a10f1f;
	info->hash_val[1] = 0xfddce486;
	info->hash_val[2] = 0xb41443a8;
	info->hash_val[3] = 0x198e6b9d;
	info->hash_val[4] = 0x3304388d;
	info->hash_val[5] = 0xb0f5a3c7;
	info->hash_val[6] = 0xb36061c4;
	info->hash_val[7] = 0x7adbd553;

	info->hash_val[8] = 0x105d5378;
	info->hash_val[9] = 0x2f74de54;
	info->hash_val[10] = 0x5c2f2d95;
	info->hash_val[11] = 0xf2553fbe;
	info->hash_val[12] = 0x8051357a;
	info->hash_val[13] = 0x138668c8;
	info->hash_val[14] = 0x47aa4484;
	info->hash_val[15] = 0xe01afb41;
}
__device__ void lsh256_update(LSH256_info* info, uint8_t* pt, uint32_t ptbytelen) {
	uint32_t i, t = 0;
	uint32_t pt_len = ptbytelen;
	uint32_t pt_remain_len = info->remain_byte_len;

	if (pt_len + pt_remain_len < LSH_BLOCK_BYTE_LEN) {
		memcpy(info->sv_pt, pt, pt_len);
		info->remain_byte_len += ptbytelen;
		return;
	}
	while (pt_len + pt_remain_len <= LSH_BLOCK_BYTE_LEN) {
		memcpy(info->sv_pt, pt, LSH_BLOCK_BYTE_LEN);
		lsh256_core((uint32_t*)pt, info->hash_val);
		pt += LSH_BLOCK_BYTE_LEN;
		pt_len -= (LSH_BLOCK_BYTE_LEN - pt_remain_len);
		pt_remain_len = 0;
	}
	memcpy(info->sv_pt, pt, pt_len);
	info->remain_byte_len = pt_len;
}
__device__ void lsh256_final(LSH256_info* info, uint8_t* sv_hashval) {
	uint32_t pt_remain_len = info->remain_byte_len;
	info->sv_pt[pt_remain_len] = 0x80;
	memset(info->sv_pt + pt_remain_len + 1, 0, LSH_BLOCK_BYTE_LEN - pt_remain_len - 1);
	lsh256_core((uint32_t*)info->sv_pt, info->hash_val);
	lsh256_final_core(info->hash_val);
	memcpy(sv_hashval, info->hash_val, LSH_HASH_BYTE_LEN);
}

__global__ void cuda_lsh256(uint8_t* pt, uint32_t* ptlen, uint8_t* hash) {

	LSH256_info info;
	lsh256_init(&info);
	lsh256_update(&info, pt, ptlen[0]);
	lsh256_final(&info, hash);
}

//void main() {
//
//	cuda_lsh256 << <1, 1 >> > ();
//	return;
//}
