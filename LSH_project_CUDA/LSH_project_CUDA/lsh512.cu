#include "hip/hip_runtime.h"
#include "lsh512.cuh"

__device__ void lsh512_core(uint64_t* state, uint64_t* hash) {
	asm("{\n\t"
		//Setting parameters
		".reg.u64               e_l<8>;		\n\t"
		".reg.u64               e_r<8>;		\n\t"
		".reg.u64               o_l<8>;		\n\t"
		".reg.u64               o_r<8>;		\n\t"
		".reg.u64               cv_l<8>;	\n\t"
		".reg.u64               cv_r<8>;	\n\t"
		".reg.u64				tmp<1>;		\n\t"

		//hash value copy
		"mov.u64 	            cv_l0,        %16;	\n\t"
		"mov.u64 	            cv_l1,        %17;	\n\t"
		"mov.u64 	            cv_l2,        %18;	\n\t"
		"mov.u64 	            cv_l3,        %19;	\n\t"
		"mov.u64 	            cv_l4,        %20;	\n\t"
		"mov.u64 	            cv_l5,        %21;	\n\t"
		"mov.u64 	            cv_l6,        %22;	\n\t"
		"mov.u64 	            cv_l7,        %23;	\n\t"

		"mov.u64 	            cv_r0,        %24;	\n\t"
		"mov.u64 	            cv_r1,        %25;	\n\t"
		"mov.u64 	            cv_r2,        %26;	\n\t"
		"mov.u64 	            cv_r3,        %27;	\n\t"
		"mov.u64 	            cv_r4,        %28;	\n\t"
		"mov.u64 	            cv_r5,        %29;	\n\t"
		"mov.u64 	            cv_r6,        %30;	\n\t"
		"mov.u64 	            cv_r7,        %31;	\n\t"
		//data value copy
		"mov.u64 	            e_l0,			%32;	\n\t"
		"mov.u64 	            e_l1,			%33;	\n\t"
		"mov.u64 	            e_l2,			%34;	\n\t"
		"mov.u64 	            e_l3,			%35;	\n\t"
		"mov.u64 	            e_l4,			%36;	\n\t"
		"mov.u64 	            e_l5,			%37;	\n\t"
		"mov.u64 	            e_l6,			%38;	\n\t"
		"mov.u64 	            e_l7,			%39;	\n\t"
		"mov.u64 	            e_r0,			%40;	\n\t"
		"mov.u64 	            e_r1,			%41;	\n\t"
		"mov.u64 	            e_r2,			%42;	\n\t"
		"mov.u64 	            e_r3,			%43;	\n\t"
		"mov.u64 	            e_r4,			%44;	\n\t"
		"mov.u64 	            e_r5,			%45;	\n\t"
		"mov.u64 	            e_r6,			%46;	\n\t"
		"mov.u64 	            e_r7,			%47;	\n\t"
		"mov.u64 	            o_l0,			%48;	\n\t"
		"mov.u64 	            o_l1,			%49;	\n\t"
		"mov.u64 	            o_l2,			%50;	\n\t"
		"mov.u64 	            o_l3,			%51;	\n\t"
		"mov.u64 	            o_l4,			%52;	\n\t"
		"mov.u64 	            o_l5,			%53;	\n\t"
		"mov.u64 	            o_l6,			%54;	\n\t"
		"mov.u64 	            o_l7,			%55;	\n\t"
		"mov.u64 	            o_r0,			%56;	\n\t"
		"mov.u64 	            o_r1,			%57;	\n\t"
		"mov.u64 	            o_r2,			%58;	\n\t"
		"mov.u64 	            o_r3,			%59;	\n\t"
		"mov.u64 	            o_r4,			%60;	\n\t"
		"mov.u64 	            o_r5,			%61;	\n\t"
		"mov.u64 	            o_r6,			%62;	\n\t"
		"mov.u64 	            o_r7,			%63;	\n\t"

		//! MSG_ADD_EVEN
		"xor.b64				cv_l0, e_l0, cv_l0;	\n\t"
		"xor.b64				cv_l1, e_l1, cv_l1;	\n\t"
		"xor.b64				cv_l2, e_l2, cv_l2;	\n\t"
		"xor.b64				cv_l3, e_l3, cv_l3;	\n\t"
		"xor.b64				cv_l4, e_l4, cv_l4;	\n\t"
		"xor.b64				cv_l5, e_l5, cv_l5;	\n\t"
		"xor.b64				cv_l6, e_l6, cv_l6;	\n\t"
		"xor.b64				cv_l7, e_l7, cv_l7;	\n\t"
		"xor.b64				cv_r0, e_r0, cv_r0;	\n\t"
		"xor.b64				cv_r1, e_r1, cv_r1;	\n\t"
		"xor.b64				cv_r2, e_r2, cv_r2;	\n\t"
		"xor.b64				cv_r3, e_r3, cv_r3;	\n\t"
		"xor.b64				cv_r4, e_r4, cv_r4;	\n\t"
		"xor.b64				cv_r5, e_r5, cv_r5;	\n\t"
		"xor.b64				cv_r6, e_r6, cv_r6;	\n\t"
		"xor.b64				cv_r7, e_r7, cv_r7;	\n\t"
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0x97884283c938982a;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xba1fca93533e2355;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xc519a2e87aeb1c03;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x9a0fc95462af17b1;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xfc3dda8ab019a82b;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x02825d079a895407;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x79f2d0a7ee06a6f7;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xd76d15eed9fdf5fe;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x1fcac64d01d0c2c1;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xd9ea5de69161790f;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xdebc8b6366071fc8;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xa9d91db711c6c94b;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x3a18653ac9c1d427;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x84df64a223dd5b09;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x6cc37895f4ad9e70;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x448304c8d7f3f4d5;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!loop start
		//!MsgExp
		"mov.b64 	            tmp0, e_l0;\n\t"
		"add.u64				e_l0, o_l0, e_l3;\n\t"
		"add.u64				e_l3, o_l3, e_l1;\n\t"
		"add.u64				e_l1, o_l1, e_l2;\n\t"
		"add.u64				e_l2, o_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, e_l4;\n\t"
		"add.u64				e_l4, o_l4, e_l7;\n\t"
		"add.u64				e_l7, o_l7, e_l6;\n\t"
		"add.u64				e_l6, o_l6, e_l5;\n\t"
		"add.u64				e_l5, o_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, e_r0;\n\t"
		"add.u64				e_r0, o_r0, e_r3;\n\t"
		"add.u64				e_r3, o_r3, e_r1;\n\t"
		"add.u64				e_r1, o_r1, e_r2;\n\t"
		"add.u64				e_r2, o_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, e_r4;\n\t"
		"add.u64				e_r4, o_r4, e_r7;\n\t"
		"add.u64				e_r7, o_r7, e_r6;\n\t"
		"add.u64				e_r6, o_r6, e_r5;\n\t"
		"add.u64				e_r5, o_r5, tmp0;\n\t"
		//! msg_add_even(i_state)
		"xor.b64				cv_l0, e_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, e_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, e_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, e_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, e_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, e_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, e_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, e_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, e_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, e_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, e_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, e_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, e_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, e_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, e_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, e_r7, cv_r7;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0xea91134ed29383e0;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xc4484477f2da88e8;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x9b47eec96d26e8a6;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x82f6d4c8d89014f4;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x527da0048b95fb61;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x644406c60138648d;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x303c0e8aa24c0edc;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xc787cda0cbe8ca19;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x7ba46221661764ca;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x0c8cbc6acd6371ac;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xe336b836940f8f41;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x79cb9da168a50976;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xd01da49021915cb3;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xa84accc7399cf1f1;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x6c4a992cee5aeb0c;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x4f556e6cb4b2e3e0;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0x200683877d7c2f45;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x9949273830d51db8;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x19eeeecaa39ed124;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x45693f0a0dae7fef;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xedc234b1b2ee1083;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xf3179400d68ee399;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xb6e3c61b4945f778;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xa4c3db216796c42f;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x268a0b04f9ab7465;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xe2705f6905f2d651;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x08ddb96e426ff53d;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xaea84917bc2e6f34;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xaff6e664a0fe9470;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x0aab94d765727d8c;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x9aa9e1648f3d702e;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x689efc88fe5af3d3;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0xb0950ffea51fd98b;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x52cfc86ef8c92833;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xe69727b0b2653245;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x56f160d3ea9da3e2;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xa6dd4b059f93051f;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xb6406c3cd7f00996;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x448b45f3ccad9ec8;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x079b8587594ec73b;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x45a50ea3c4f9653b;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x22983767c1f15b85;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x7dbed8631797782b;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x485234be88418638;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x842850a5329824c5;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xf6aca914c7f9a04c;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xcfd139c07a4c670c;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xa3210ce0a8160242;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0xeab3b268be5ea080;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xbacf9f29b34ce0a7;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x3c973b7aaf0fa3a8;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x9a86f346c9c7be80;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xac78f5d7cabcea49;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xa355bddcc199ed42;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xa10afa3ac6b373db;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xc42ded88be1844e5;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x9e661b271cff216a;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x8a6ec8dd002d8861;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xd3d2b629beb34be4;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x217a3a1091863f1a;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x256ecda287a733f5;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xf9139a9e5b872fe5;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xac0535017a274f7c;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xf21b7646d65d2aa9;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0x048142441c208c08;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xf937a5dd2db5e9eb;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xa688dfe871ff30b7;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x9bb44aa217c5593b;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x943c702a2edb291a;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x0cae38f9e2b715de;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xb13a367ba176cc28;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x0d91bd1d3387d49b;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x85c386603cac940c;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x30dd830ae39fd5e4;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x2f68c85a712fe85d;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x4ffeecb9dd1e94d6;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xd0ac9a590a0443ae;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xbae732dc99ccf3ea;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xeb70b21d1842f4d9;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x9f4eda50bb5c6fa8;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0x4949e69ce940a091;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x0e608dee8375ba14;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x983122cba118458c;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x4eeba696fbb36b25;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x7d46f3630e47f27e;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xa21a0f7666c0dea4;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x5c22cf355b37cec4;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xee292b0c17cc1847;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x9330838629e131da;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x6eee7c71f92fce22;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xc953ee6cb95dd224;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x3a923d92af1e9073;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xc43a5671563a70fb;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xbc2985dd279f8346;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x7ef2049093069320;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x17543723e3e46035;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0xc3b409b00b130c6d;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x5d6aee6b28fdf090;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x1d425b26172ff6ed;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xcccfd041cdaf03ad;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xfe90c7c790ab6cbf;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xe5af6304c722ca02;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x70f695239999b39e;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x6b8b5b07c844954c;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x77bdb9bb1e1f7a30;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xc859599426ee80ed;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x5f9d813d4726e40a;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x9ca0120f7cb2b179;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x8f588f583c182cbd;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x951267cbe9eccce7;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x678bb8bd334d520e;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xf6e662d00cd9e1b7;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0x357774d93d99aaa7;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x21b2edbb156f6eb5;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xfd1ebe846e0aee69;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x3cb2218c2f642b15;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xe7e7e7945444ea4c;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xa77a33b5d6b9b47c;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xf34475f0809f6075;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xdd4932dce6bb99ad;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0xacec4e16d74451dc;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xd4a0a8d084de23d6;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x1bdd42f278f95866;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xeed3adbb938f4051;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xcfcf7be8992f3733;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x21ade98c906e3123;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x37ba66711fffd668;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x267c0fc3a255478a;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0x993a64ee1b962e88;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x754979556301faaa;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xf920356b7251be81;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xc281694f22cf923f;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x9f4b6481c8666b02;\n\t"
		"xor.b64				cv_l5, cv_l5, 0xcf97761cfe9f5444;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xf220d7911fd63e9f;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xa28bd365f79cd1b0;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0xd39f5309b1c4b721;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xbec2ceb864fca51f;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x1955a0ddc410407a;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x43eab871f261d201;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xeaafe64a2ed16da1;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x670d931b9df39913;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x12f868b0f614de91;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x2e5f395d946e8252;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0x72f25cbb767bd8f4;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x8191871d61a1c4dd;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x6ef67ea1d450ba93;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x2ea32a645433d344;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x9a963079003f0f8b;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x74a0aeb9918cac7a;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x0b6119a70af36fa3;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x8d9896f202f0d480;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x654f1831f254cd66;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x1318a47f0366a25e;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x65752076250b4e01;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xd1cd8eb888071772;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x30c6a9793f4e9b25;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x154f684b1e3926ee;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x6c7ac0b1fe6312ae;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x262f88f4f3c5550d;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0xb4674a24472233cb;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x2bbd23826a090071;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xda95969b30594f66;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x9f5c47408f1e8a43;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xf77022b88de9c055;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x64b7b36957601503;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xe73b72b06175c11a;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x55b87de8b91a6233;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x1bb16e6b6955ff7f;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xe8e0a5ec7309719c;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x702c31cb89a8b640;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xfba387cfada8cde2;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x6792db4677aa164c;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x1c6b1cc0b7751867;\n\t"
		"xor.b64				cv_l6, cv_l6, 0x22ae2311d736dc01;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x0e3666a1d37c9588;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0xcd1fd9d4bf557e9a;\n\t"
		"xor.b64				cv_l1, cv_l1, 0xc986925f7c7b0e84;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x9c5dfd55325ef6b0;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x9f2b577d5676b0dd;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xfa6e21be21c062b3;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x8787dd782c8d7f83;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xd0d134e90e12dd23;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x449d087550121d96;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0xecf9ae9414d41967;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x5018f1dbf789934d;\n\t"
		"xor.b64				cv_l2, cv_l2, 0xfa5b52879155a74c;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xca82d4d3cd278e7c;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x688fdfdfe22316ad;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x0f6555a4ba0d030a;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xa2061df720f000f3;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xe1a57dc5622fb3da;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//MIX(even)
		//rotate_blk(cv_l, 23)
		"shl.b64				tmp0,	cv_l0,	23;\n\t"
		"shr.b64				cv_l0,	cv_l0,	41;\n\t"
		"or.b64					cv_l0,	tmp0,	cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	23;\n\t"
		"shr.b64				cv_l1,	cv_l1,	41;\n\t"
		"or.b64					cv_l1,	tmp0,	cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	23;\n\t"
		"shr.b64				cv_l2,	cv_l2,	41;\n\t"
		"or.b64					cv_l2,	tmp0,	cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	23;\n\t"
		"shr.b64				cv_l3,	cv_l3,	41;\n\t"
		"or.b64					cv_l3,	tmp0,	cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	23;\n\t"
		"shr.b64				cv_l4,	cv_l4,	41;\n\t"
		"or.b64					cv_l4,	tmp0,	cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	23;\n\t"
		"shr.b64				cv_l5,	cv_l5,	41;\n\t"
		"or.b64					cv_l5,	tmp0,	cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	23;\n\t"
		"shr.b64				cv_l6,	cv_l6,	41;\n\t"
		"or.b64					cv_l6,	tmp0,	cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	23;\n\t"
		"shr.b64				cv_l7,	cv_l7,	41;\n\t"
		"or.b64					cv_l7,	tmp0,	cv_l7;\n\t"
		//xor_with_const
		"xor.b64				cv_l0, cv_l0, 0xe6a842a8e8ed8153;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x690acdd3811ce09d;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x55adda18e6fcf446;\n\t"
		"xor.b64				cv_l3, cv_l3, 0x4d57a8a0f4b60b46;\n\t"
		"xor.b64				cv_l4, cv_l4, 0xf86fbfc20539c415;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x74bafa5ec7100d19;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xa824151810f0f495;\n\t"
		"xor.b64				cv_l7, cv_l7, 0x8723432791e38ebb;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 59)
		"shl.b64				tmp0,	cv_r0,	59;\n\t"
		"shr.b64				cv_r0,	cv_r0,	5;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	59;\n\t"
		"shr.b64				cv_r1,	cv_r1,	5;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	59;\n\t"
		"shr.b64				cv_r2,	cv_r2,	5;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	59;\n\t"
		"shr.b64				cv_r3,	cv_r3,	5;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	59;\n\t"
		"shr.b64				cv_r4,	cv_r4,	5;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	59;\n\t"
		"shr.b64				cv_r5,	cv_r5,	5;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	59;\n\t"
		"shr.b64				cv_r6,	cv_r6,	5;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	59;\n\t"
		"shr.b64				cv_r7,	cv_r7,	5;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"
		//!MsgExp(odd)
		"mov.b64 	            tmp0, o_l0;\n\t"
		"add.u64				o_l0, e_l0, o_l3;\n\t"
		"add.u64				o_l3, e_l3, o_l1;\n\t"
		"add.u64				o_l1, e_l1, o_l2;\n\t"
		"add.u64				o_l2, e_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_l4;\n\t"
		"add.u64				o_l4, e_l4, o_l7;\n\t"
		"add.u64				o_l7, e_l7, o_l6;\n\t"
		"add.u64				o_l6, e_l6, o_l5;\n\t"
		"add.u64				o_l5, e_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r0;\n\t"
		"add.u64				o_r0, e_r0, o_r3;\n\t"
		"add.u64				o_r3, e_r3, o_r1;\n\t"
		"add.u64				o_r1, e_r1, o_r2;\n\t"
		"add.u64				o_r2, e_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, o_r4;\n\t"
		"add.u64				o_r4, e_r4, o_r7;\n\t"
		"add.u64				o_r7, e_r7, o_r6;\n\t"
		"add.u64				o_r6, e_r6, o_r5;\n\t"
		"add.u64				o_r5, e_r5, tmp0;\n\t"
		//!msg_add_odd(cv_l, cv_r, i_state)
		"xor.b64				cv_l0, o_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, o_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, o_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, o_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, o_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, o_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, o_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, o_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, o_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, o_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, o_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, o_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, o_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, o_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, o_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, o_r7, cv_r7;\n\t"
		//!MIX(odd)
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_l, 7)
		"shl.b64				tmp0,	cv_l0,	7;\n\t"
		"shr.b64				cv_l0,	cv_l0,	57;\n\t"
		"or.b64					cv_l0,  tmp0, cv_l0;\n\t"
		"shl.b64				tmp0,	cv_l1,	7;\n\t"
		"shr.b64				cv_l1,	cv_l1,	57;\n\t"
		"or.b64					cv_l1,  tmp0, cv_l1;\n\t"
		"shl.b64				tmp0,	cv_l2,	7;\n\t"
		"shr.b64				cv_l2,	cv_l2,	57;\n\t"
		"or.b64					cv_l2,  tmp0, cv_l2;\n\t"
		"shl.b64				tmp0,	cv_l3,	7;\n\t"
		"shr.b64				cv_l3,	cv_l3,	57;\n\t"
		"or.b64					cv_l3,  tmp0, cv_l3;\n\t"
		"shl.b64				tmp0,	cv_l4,	7;\n\t"
		"shr.b64				cv_l4,	cv_l4,	57;\n\t"
		"or.b64					cv_l4,  tmp0, cv_l4;\n\t"
		"shl.b64				tmp0,	cv_l5,	7;\n\t"
		"shr.b64				cv_l5,	cv_l5,	57;\n\t"
		"or.b64					cv_l5,  tmp0, cv_l5;\n\t"
		"shl.b64				tmp0,	cv_l6,	7;\n\t"
		"shr.b64				cv_l6,	cv_l6,	57;\n\t"
		"or.b64					cv_l6,  tmp0, cv_l6;\n\t"
		"shl.b64				tmp0,	cv_l7,	7;\n\t"
		"shr.b64				cv_l7,	cv_l7,	57;\n\t"
		"or.b64					cv_l7,  tmp0, cv_l7;\n\t"
		//xor with const
		"xor.b64				cv_l0, cv_l0, 0x8eeaeb91d66ed539;\n\t"
		"xor.b64				cv_l1, cv_l1, 0x73d8a1549dfd7e06;\n\t"
		"xor.b64				cv_l2, cv_l2, 0x0387f2ffe3f13a9b;\n\t"
		"xor.b64				cv_l3, cv_l3, 0xa5004995aac15193;\n\t"
		"xor.b64				cv_l4, cv_l4, 0x682f81c73efdda0d;\n\t"
		"xor.b64				cv_l5, cv_l5, 0x2fb55925d71d268d;\n\t"
		"xor.b64				cv_l6, cv_l6, 0xcc392d2901e58a3d;\n\t"
		"xor.b64				cv_l7, cv_l7, 0xaa666ab975724a42;\n\t"
		//add_blk(cv_r, cv_l)
		"add.u64				cv_r0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_r1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_r2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_r3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_r4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_r5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_r6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_r7, cv_l7, cv_r7;\n\t"
		//rotate_blk(cv_r, 3)
		"shl.b64				tmp0,	cv_r0,	3;\n\t"
		"shr.b64				cv_r0,	cv_r0,	61;\n\t"
		"or.b64					cv_r0,	tmp0,	cv_r0;\n\t"
		"shl.b64				tmp0,	cv_r1,	3;\n\t"
		"shr.b64				cv_r1,	cv_r1,	61;\n\t"
		"or.b64					cv_r1,	tmp0,	cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	3;\n\t"
		"shr.b64				cv_r2,	cv_r2,	61;\n\t"
		"or.b64					cv_r2,	tmp0,	cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	3;\n\t"
		"shr.b64				cv_r3,	cv_r3,	61;\n\t"
		"or.b64					cv_r3,	tmp0,	cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	3;\n\t"
		"shr.b64				cv_r4,	cv_r4,	61;\n\t"
		"or.b64					cv_r4,	tmp0,	cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	3;\n\t"
		"shr.b64				cv_r5,	cv_r5,	61;\n\t"
		"or.b64					cv_r5,	tmp0,	cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	3;\n\t"
		"shr.b64				cv_r6,	cv_r6,	61;\n\t"
		"or.b64					cv_r6,	tmp0,	cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	3;\n\t"
		"shr.b64				cv_r7,	cv_r7,	61;\n\t"
		"or.b64					cv_r7,	tmp0,	cv_r7;\n\t"
		//add_blk(cv_l, cv_r)
		"add.u64				cv_l0, cv_l0, cv_r0;\n\t"
		"add.u64				cv_l1, cv_l1, cv_r1;\n\t"
		"add.u64				cv_l2, cv_l2, cv_r2;\n\t"
		"add.u64				cv_l3, cv_l3, cv_r3;\n\t"
		"add.u64				cv_l4, cv_l4, cv_r4;\n\t"
		"add.u64				cv_l5, cv_l5, cv_r5;\n\t"
		"add.u64				cv_l6, cv_l6, cv_r6;\n\t"
		"add.u64				cv_l7, cv_l7, cv_r7;\n\t"
		//rotate_msg_gamma(cv_r)
		"shl.b64				tmp0,	cv_r1,	16;\n\t"
		"shr.b64				cv_r1,	cv_r1,	48;\n\t"
		"or.b64					cv_r1,  tmp0, cv_r1;\n\t"
		"shl.b64				tmp0,	cv_r2,	32;\n\t"
		"shr.b64				cv_r2,	cv_r2,	32;\n\t"
		"or.b64					cv_r2,  tmp0, cv_r2;\n\t"
		"shl.b64				tmp0,	cv_r3,	48;\n\t"
		"shr.b64				cv_r3,	cv_r3,	16;\n\t"
		"or.b64					cv_r3,  tmp0, cv_r3;\n\t"
		"shl.b64				tmp0,	cv_r4,	8;\n\t"
		"shr.b64				cv_r4,	cv_r4,	56;\n\t"
		"or.b64					cv_r4,  tmp0, cv_r4;\n\t"
		"shl.b64				tmp0,	cv_r5,	24;\n\t"
		"shr.b64				cv_r5,	cv_r5,	40;\n\t"
		"or.b64					cv_r5,  tmp0, cv_r5;\n\t"
		"shl.b64				tmp0,	cv_r6,	40;\n\t"
		"shr.b64				cv_r6,	cv_r6,	24;\n\t"
		"or.b64					cv_r6,  tmp0, cv_r6;\n\t"
		"shl.b64				tmp0,	cv_r7,	56;\n\t"
		"shr.b64				cv_r7,	cv_r7,	8;\n\t"
		"or.b64					cv_r7,  tmp0, cv_r7;\n\t"
		//!word_perm(cv_l, cv_r)
		"mov.u64 	            tmp0, cv_l0;\n\t"
		"mov.u64 	            cv_l0, cv_l6;\n\t"
		"mov.u64 	            cv_l6, cv_r6;\n\t"
		"mov.u64 	            cv_r6, cv_r2;\n\t"
		"mov.u64 	            cv_r2, cv_l1;\n\t"
		"mov.u64 	            cv_l1, cv_l4;\n\t"
		"mov.u64 	            cv_l4, cv_r4;\n\t"
		"mov.u64 	            cv_r4, cv_r0;\n\t"
		"mov.u64 	            cv_r0, cv_l2;\n\t"
		"mov.u64 	            cv_l2, cv_l5;\n\t"
		"mov.u64 	            cv_l5, cv_r7;\n\t"
		"mov.u64 	            cv_r7, cv_r1;\n\t"
		"mov.u64 	            cv_r1, tmp0;\n\t"
		"mov.u64 	            tmp0, cv_l3;\n\t"
		"mov.u64 	            cv_l3, cv_l7;\n\t"
		"mov.u64 	            cv_l7, cv_r5;\n\t"
		"mov.u64 	            cv_r5, cv_r3;\n\t"
		"mov.u64 	            cv_r3, tmp0;\n\t"

		//!msg_exp_even
		"mov.b64 	            tmp0, e_l0;\n\t"
		"add.u64				e_l0, o_l0, e_l3;\n\t"
		"add.u64				e_l3, o_l3, e_l1;\n\t"
		"add.u64				e_l1, o_l1, e_l2;\n\t"
		"add.u64				e_l2, o_l2, tmp0;\n\t"
		"mov.b64 	            tmp0, e_l4;\n\t"
		"add.u64				e_l4, o_l4, e_l7;\n\t"
		"add.u64				e_l7, o_l7, e_l6;\n\t"
		"add.u64				e_l6, o_l6, e_l5;\n\t"
		"add.u64				e_l5, o_l5, tmp0;\n\t"
		"mov.b64 	            tmp0, e_r0;\n\t"
		"add.u64				e_r0, o_r0, e_r3;\n\t"
		"add.u64				e_r3, o_r3, e_r1;\n\t"
		"add.u64				e_r1, o_r1, e_r2;\n\t"
		"add.u64				e_r2, o_r2, tmp0;\n\t"
		"mov.b64 	            tmp0, e_r4;\n\t"
		"add.u64				e_r4, o_r4, e_r7;\n\t"
		"add.u64				e_r7, o_r7, e_r6;\n\t"
		"add.u64				e_r6, o_r6, e_r5;\n\t"
		"add.u64				e_r5, o_r5, tmp0;\n\t"

		//! msg_add_even(i_state)
		"xor.b64				cv_l0, e_l0, cv_l0;\n\t"
		"xor.b64				cv_l1, e_l1, cv_l1;\n\t"
		"xor.b64				cv_l2, e_l2, cv_l2;\n\t"
		"xor.b64				cv_l3, e_l3, cv_l3;\n\t"
		"xor.b64				cv_l4, e_l4, cv_l4;\n\t"
		"xor.b64				cv_l5, e_l5, cv_l5;\n\t"
		"xor.b64				cv_l6, e_l6, cv_l6;\n\t"
		"xor.b64				cv_l7, e_l7, cv_l7;\n\t"
		"xor.b64				cv_r0, e_r0, cv_r0;\n\t"
		"xor.b64				cv_r1, e_r1, cv_r1;\n\t"
		"xor.b64				cv_r2, e_r2, cv_r2;\n\t"
		"xor.b64				cv_r3, e_r3, cv_r3;\n\t"
		"xor.b64				cv_r4, e_r4, cv_r4;\n\t"
		"xor.b64				cv_r5, e_r5, cv_r5;\n\t"
		"xor.b64				cv_r6, e_r6, cv_r6;\n\t"
		"xor.b64				cv_r7, e_r7, cv_r7;\n\t"
		"mov.b64 	            %0,  cv_l0;\n\t"
		"mov.b64 	            %1,  cv_l1;\n\t"
		"mov.b64 	            %2,  cv_l2;\n\t"
		"mov.b64 	            %3,  cv_l3;\n\t"
		"mov.b64 	            %4,  cv_l4;\n\t"
		"mov.b64 	            %5,  cv_l5;\n\t"
		"mov.b64 	            %6,  cv_l6;\n\t"
		"mov.b64 	            %7,  cv_l7;\n\t"

		"mov.b64 	            %8,  cv_r0;\n\t"
		"mov.b64 	            %9,  cv_r1;\n\t"
		"mov.b64 	            %10,  cv_r2;\n\t"
		"mov.b64 	            %11,  cv_r3;\n\t"
		"mov.b64 	            %12,  cv_r4;\n\t"
		"mov.b64 	            %13,  cv_r5;\n\t"
		"mov.b64 	            %14,  cv_r6;\n\t"
		"mov.b64 	            %15,  cv_r7;\n\t"

		"}"
		:	"+l"(hash[0]), "+l"(hash[1]), "+l"(hash[2]), "+l"(hash[3]), "+l"(hash[4]), "+l"(hash[5]), "+l"(hash[6]), "+l"(hash[7]),
		"+l"(hash[8]), "+l"(hash[9]), "+l"(hash[10]), "+l"(hash[11]), "+l"(hash[12]), "+l"(hash[13]), "+l"(hash[14]), "+l"(hash[15])
		: "l"(hash[0]), "l"(hash[1]), "l"(hash[2]), "l"(hash[3]), "l"(hash[4]), "l"(hash[5]), "l"(hash[6]), "l"(hash[7]),
		"l"(hash[8]), "l"(hash[9]), "l"(hash[10]), "l"(hash[11]), "l"(hash[12]), "l"(hash[13]), "l"(hash[14]), "l"(hash[15])
		"l"(state[0]), "l"(state[1]), "l"(state[2]), "l"(state[3]), "l"(state[4]), "l"(state[5]), "l"(state[6]), "l"(state[7]),
		"l"(state[8]), "l"(state[9]), "l"(state[10]), "l"(state[11]), "l"(state[12]), "l"(state[13]), "l"(state[14]), "l"(state[15]),
		"l"(state[16]), "l"(state[17]), "l"(state[18]), "l"(state[19]), "l"(state[20]), "l"(state[21]), "l"(state[22]), "l"(state[23]),
		"l"(state[24]), "l"(state[25]), "l"(state[26]), "l"(state[27]), "l"(state[28]), "l"(state[29]), "l"(state[30]), "l"(state[31])
	);
}

__device__ void lsh512_final_core(uint64_t* hash_val) {
	hash_val[0] ^= hash_val[8];
	hash_val[1] ^= hash_val[9];
	hash_val[2] ^= hash_val[10];
	hash_val[3] ^= hash_val[11];
	hash_val[4] ^= hash_val[12];
	hash_val[5] ^= hash_val[13];
	hash_val[6] ^= hash_val[14];
	hash_val[7] ^= hash_val[15];

}
__device__ void lsh512_init(LSH512_info* info) {
	for (int i = 0; i < LSH_BLOCK_BYTE_LEN; i++)
		info->sv_pt[i] = 0;
	info->remain_byte_len = 0;

	info->hash_val[0] = 0xadd50f3c7f07094e;
	info->hash_val[1] = 0xe3f3cee8f9418a4f;
	info->hash_val[2] = 0xb527ecde5b3d0ae9;
	info->hash_val[3] = 0x2ef6dec68076f501;
	info->hash_val[4] = 0x8cb994cae5aca216;
	info->hash_val[5] = 0xfbb9eae4bba48cc7;
	info->hash_val[6] = 0x650a526174725fea;
	info->hash_val[7] = 0x1f9a61a73f8d8085;

	info->hash_val[8] = 0xb6607378173b539b;
	info->hash_val[9] = 0x1bc99853b0c0b9ed;
	info->hash_val[10] = 0xdf727fc19b182d47;
	info->hash_val[11] = 0xdbef360cf893a457;
	info->hash_val[12] = 0x4981f5e570147e80;
	info->hash_val[13] = 0xd00c4490ca7d3e30;
	info->hash_val[14] = 0x5d73940c0e4ae1ec;
	info->hash_val[15] = 0x894085e2edb2d819;
}

//test parameters
#define LSH_PT_SIZE		128
#define LSH_HASH_SIZE	64
#define streams			32
__device__ void lsh512_update(LSH512_info* info, uint8_t* pt, uint32_t ptbytelen) {
	uint32_t i, t = 0;
	uint32_t pt_len = ptbytelen;
	uint32_t pt_remain_len = info->remain_byte_len;
	uint8_t test_sv_pt[LSH_PT_SIZE] = { 0, };

	for (i = 0; i < LSH_PT_SIZE; i++) {
		test_sv_pt[i] = pt[i];
	}
	i = 0;

	if (pt_len + pt_remain_len < LSH_BLOCK_BYTE_LEN) {
		memcpy(info->sv_pt + pt_remain_len, test_sv_pt, pt_len);
		info->remain_byte_len += ptbytelen;
		return;
	}


	while (pt_len + pt_remain_len >= LSH_BLOCK_BYTE_LEN) {
		memcpy(info->sv_pt, test_sv_pt + i * LSH_BLOCK_BYTE_LEN, LSH_BLOCK_BYTE_LEN);
		lsh512_core((uint64_t*)info->sv_pt, info->hash_val);
		i++;
		pt_len -= (LSH_BLOCK_BYTE_LEN - pt_remain_len);
		pt_remain_len = 0;
	}

	memcpy(info->sv_pt, test_sv_pt + i * LSH_BLOCK_BYTE_LEN, pt_len);
	info->remain_byte_len = pt_len;
}

__device__ void lsh512_final(LSH512_info* info, uint8_t* sv_hashval) {
	uint32_t pt_remain_len = info->remain_byte_len;
	info->sv_pt[pt_remain_len] = 0x80;
	memset(info->sv_pt + pt_remain_len + 1, 0, LSH_BLOCK_BYTE_LEN - pt_remain_len - 1);
	lsh512_core((uint64_t*)info->sv_pt, info->hash_val);
	lsh512_final_core(info->hash_val);
	memcpy(sv_hashval, info->hash_val, LSH_HASH_BYTE_LEN);
}


__global__ void cuda_lsh512(uint8_t* pt, uint8_t* hash) {

	uint64_t tid_index = (blockDim.x * blockIdx.x) + threadIdx.x;
	uint64_t memory_index = (gridDim.x * blockDim.x);

	uint8_t buffer[LSH_PT_SIZE] = { 0, };
	uint8_t tmp[64] = { 0, };


	//ptx
	//LSH512_info info;
	//lsh512_init(&info);
	//lsh512_update(&info, pt + LSH_PT_SIZE * tid_index, LSH_PT_SIZE);
	//lsh512_final(&info, hash + LSH_HASH_SIZE * tid_index);


	//ptx + coalesced memory

	for (int i = 0; i < LSH_PT_SIZE; i++) {
		buffer[i] = pt[tid_index + i * memory_index];
	}

	LSH512_info info;
	lsh512_init(&info);
	lsh512_update(&info, buffer, LSH_PT_SIZE);
	lsh512_final(&info, tmp);


	for (int i = 0; i < 64; i++) {
		hash[tid_index + i * memory_index] = tmp[i];
	}

}

void ptx_performance_test(uint32_t blocksize, uint32_t threadsize) {
	uint8_t* cpu_pt = NULL;
	uint8_t* cpu_ha = NULL;
	uint8_t* gpu_pt = NULL;
	uint8_t* gpu_ha = NULL;

	hipEvent_t start, stop;
	hipError_t err;
	float elapsed_time_ms = 0.0f;

	cpu_pt = (uint8_t*)malloc(sizeof(uint8_t) * blocksize * threadsize * LSH_PT_SIZE);
	if (cpu_pt == NULL) {
		printf("cpu_pt malloc is fail\n");
		return;
	}
	cpu_ha = (uint8_t*)malloc(sizeof(uint8_t) * blocksize * threadsize * LSH_HASH_SIZE);
	if (cpu_pt == NULL) {
		printf("cpu_pt malloc is fail\n");
		return;
	}
	err = hipMalloc((void**)&gpu_pt, sizeof(uint8_t) * blocksize * threadsize * LSH_PT_SIZE);
	if (err != hipSuccess) {
		printf("gpu_pt malloc faile\n");
		return;
	}
	err = hipMalloc((void**)&gpu_ha, sizeof(uint8_t) * blocksize * threadsize * LSH_HASH_SIZE);
	if (err != hipSuccess) {
		printf("gpu_ha malloc faile\n");
		return;
	}

	for (int i = 0; i < blocksize * threadsize * LSH_PT_SIZE; i++) {
		cpu_pt[i] = ((i * i + i) * i + i) & 0xff;
	}

	err = hipMemcpy(gpu_pt, cpu_pt, sizeof(uint8_t) * blocksize * threadsize * LSH_PT_SIZE, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy(cpu_pt -> gpu_pt) is fail\n");
		return;
	}

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int x = 0; x < 1000; x++) {
		hipMemcpy(gpu_pt, cpu_pt, sizeof(uint8_t) * blocksize * threadsize * LSH_PT_SIZE, hipMemcpyHostToDevice);
		cuda_lsh512 << <blocksize, threadsize >> > (gpu_pt, gpu_ha);
		hipMemcpy(cpu_ha, gpu_ha, sizeof(uint8_t) * blocksize * threadsize * LSH_HASH_SIZE, hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);

	elapsed_time_ms = elapsed_time_ms / 1000;
	elapsed_time_ms = 1000 / elapsed_time_ms;
	elapsed_time_ms = blocksize * threadsize * elapsed_time_ms;
	elapsed_time_ms /= 1000;
	elapsed_time_ms /= 1000;

	printf("operation/sec (%d, %d) = %4.2fMH/s\n", blocksize, threadsize, elapsed_time_ms);
	printf("%02X\n", cpu_ha[0]);
	hipFree(gpu_pt);
	hipFree(gpu_ha);
	free(cpu_pt);
	free(cpu_ha);
}

void cudastream_test(uint32_t blocksize, uint32_t threadsize) {
	uint64_t pt_size = blocksize * threadsize * LSH_PT_SIZE;
	uint64_t ha_size = blocksize * threadsize * LSH_HASH_SIZE;
	hipStream_t S[streams];

	hipEvent_t start, stop;
	float elapsed_time_ms = 0.0f;

#if 1
	uint8_t* pt0 = NULL;
	uint8_t* pt1 = NULL;
	uint8_t* pt2 = NULL;
	uint8_t* pt3 = NULL;
	uint8_t* pt4 = NULL;
	uint8_t* pt5 = NULL;
	uint8_t* pt6 = NULL;
	uint8_t* pt7 = NULL;
	uint8_t* pt8 = NULL;
	uint8_t* pt9 = NULL;
	uint8_t* pt10 = NULL;
	uint8_t* pt11 = NULL;
	uint8_t* pt12 = NULL;
	uint8_t* pt13 = NULL;
	uint8_t* pt14 = NULL;
	uint8_t* pt15 = NULL;
	uint8_t* pt16 = NULL;
	uint8_t* pt17 = NULL;
	uint8_t* pt18 = NULL;
	uint8_t* pt19 = NULL;
	uint8_t* pt20 = NULL;
	uint8_t* pt21 = NULL;
	uint8_t* pt22 = NULL;
	uint8_t* pt23 = NULL;
	uint8_t* pt24 = NULL;
	uint8_t* pt25 = NULL;
	uint8_t* pt26 = NULL;
	uint8_t* pt27 = NULL;
	uint8_t* pt28 = NULL;
	uint8_t* pt29 = NULL;
	uint8_t* pt30 = NULL;
	uint8_t* pt31 = NULL;

	uint8_t* ha0 = NULL;
	uint8_t* ha1 = NULL;
	uint8_t* ha2 = NULL;
	uint8_t* ha3 = NULL;
	uint8_t* ha4 = NULL;
	uint8_t* ha5 = NULL;
	uint8_t* ha6 = NULL;
	uint8_t* ha7 = NULL;
	uint8_t* ha8 = NULL;
	uint8_t* ha9 = NULL;
	uint8_t* ha10 = NULL;
	uint8_t* ha11 = NULL;
	uint8_t* ha12 = NULL;
	uint8_t* ha13 = NULL;
	uint8_t* ha14 = NULL;
	uint8_t* ha15 = NULL;
	uint8_t* ha16 = NULL;
	uint8_t* ha17 = NULL;
	uint8_t* ha18 = NULL;
	uint8_t* ha19 = NULL;
	uint8_t* ha20 = NULL;
	uint8_t* ha21 = NULL;
	uint8_t* ha22 = NULL;
	uint8_t* ha23 = NULL;
	uint8_t* ha24 = NULL;
	uint8_t* ha25 = NULL;
	uint8_t* ha26 = NULL;
	uint8_t* ha27 = NULL;
	uint8_t* ha28 = NULL;
	uint8_t* ha29 = NULL;
	uint8_t* ha30 = NULL;
	uint8_t* ha31 = NULL;

	hipHostAlloc((void**)&pt0, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt1, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt2, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt3, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt4, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt5, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt6, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt7, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt8, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt9, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt10, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt11, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt12, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt13, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt14, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt15, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt16, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt17, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt18, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt19, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt20, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt21, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt22, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt23, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt24, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt25, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt26, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt27, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt28, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt29, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt30, pt_size, hipHostMallocDefault);
	hipHostAlloc((void**)&pt31, pt_size, hipHostMallocDefault);

	hipHostAlloc((void**)&ha0, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha1, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha2, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha3, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha4, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha5, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha6, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha7, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha8, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha9, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha10, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha11, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha12, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha13, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha14, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha15, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha16, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha17, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha18, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha19, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha20, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha21, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha22, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha23, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha24, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha25, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha26, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha27, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha28, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha29, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha30, ha_size, hipHostMallocDefault);
	hipHostAlloc((void**)&ha31, ha_size, hipHostMallocDefault);
#endif

#if 1
	uint8_t* gpu_pt0 = NULL;
	uint8_t* gpu_pt1 = NULL;
	uint8_t* gpu_pt2 = NULL;
	uint8_t* gpu_pt3 = NULL;
	uint8_t* gpu_pt4 = NULL;
	uint8_t* gpu_pt5 = NULL;
	uint8_t* gpu_pt6 = NULL;
	uint8_t* gpu_pt7 = NULL;
	uint8_t* gpu_pt8 = NULL;
	uint8_t* gpu_pt9 = NULL;
	uint8_t* gpu_pt10 = NULL;
	uint8_t* gpu_pt11 = NULL;
	uint8_t* gpu_pt12 = NULL;
	uint8_t* gpu_pt13 = NULL;
	uint8_t* gpu_pt14 = NULL;
	uint8_t* gpu_pt15 = NULL;
	uint8_t* gpu_pt16 = NULL;
	uint8_t* gpu_pt17 = NULL;
	uint8_t* gpu_pt18 = NULL;
	uint8_t* gpu_pt19 = NULL;
	uint8_t* gpu_pt20 = NULL;
	uint8_t* gpu_pt21 = NULL;
	uint8_t* gpu_pt22 = NULL;
	uint8_t* gpu_pt23 = NULL;
	uint8_t* gpu_pt24 = NULL;
	uint8_t* gpu_pt25 = NULL;
	uint8_t* gpu_pt26 = NULL;
	uint8_t* gpu_pt27 = NULL;
	uint8_t* gpu_pt28 = NULL;
	uint8_t* gpu_pt29 = NULL;
	uint8_t* gpu_pt30 = NULL;
	uint8_t* gpu_pt31 = NULL;

	uint8_t* gpu_ha0 = NULL;
	uint8_t* gpu_ha1 = NULL;
	uint8_t* gpu_ha2 = NULL;
	uint8_t* gpu_ha3 = NULL;
	uint8_t* gpu_ha4 = NULL;
	uint8_t* gpu_ha5 = NULL;
	uint8_t* gpu_ha6 = NULL;
	uint8_t* gpu_ha7 = NULL;
	uint8_t* gpu_ha8 = NULL;
	uint8_t* gpu_ha9 = NULL;
	uint8_t* gpu_ha10 = NULL;
	uint8_t* gpu_ha11 = NULL;
	uint8_t* gpu_ha12 = NULL;
	uint8_t* gpu_ha13 = NULL;
	uint8_t* gpu_ha14 = NULL;
	uint8_t* gpu_ha15 = NULL;
	uint8_t* gpu_ha16 = NULL;
	uint8_t* gpu_ha17 = NULL;
	uint8_t* gpu_ha18 = NULL;
	uint8_t* gpu_ha19 = NULL;
	uint8_t* gpu_ha20 = NULL;
	uint8_t* gpu_ha21 = NULL;
	uint8_t* gpu_ha22 = NULL;
	uint8_t* gpu_ha23 = NULL;
	uint8_t* gpu_ha24 = NULL;
	uint8_t* gpu_ha25 = NULL;
	uint8_t* gpu_ha26 = NULL;
	uint8_t* gpu_ha27 = NULL;
	uint8_t* gpu_ha28 = NULL;
	uint8_t* gpu_ha29 = NULL;
	uint8_t* gpu_ha30 = NULL;
	uint8_t* gpu_ha31 = NULL;

	hipMalloc((void**)&gpu_pt0, pt_size);
	hipMalloc((void**)&gpu_pt1, pt_size);
	hipMalloc((void**)&gpu_pt2, pt_size);
	hipMalloc((void**)&gpu_pt3, pt_size);
	hipMalloc((void**)&gpu_pt4, pt_size);
	hipMalloc((void**)&gpu_pt5, pt_size);
	hipMalloc((void**)&gpu_pt6, pt_size);
	hipMalloc((void**)&gpu_pt7, pt_size);
	hipMalloc((void**)&gpu_pt8, pt_size);
	hipMalloc((void**)&gpu_pt9, pt_size);
	hipMalloc((void**)&gpu_pt10, pt_size);
	hipMalloc((void**)&gpu_pt11, pt_size);
	hipMalloc((void**)&gpu_pt12, pt_size);
	hipMalloc((void**)&gpu_pt13, pt_size);
	hipMalloc((void**)&gpu_pt14, pt_size);
	hipMalloc((void**)&gpu_pt15, pt_size);
	hipMalloc((void**)&gpu_pt16, pt_size);
	hipMalloc((void**)&gpu_pt17, pt_size);
	hipMalloc((void**)&gpu_pt18, pt_size);
	hipMalloc((void**)&gpu_pt19, pt_size);
	hipMalloc((void**)&gpu_pt20, pt_size);
	hipMalloc((void**)&gpu_pt21, pt_size);
	hipMalloc((void**)&gpu_pt22, pt_size);
	hipMalloc((void**)&gpu_pt23, pt_size);
	hipMalloc((void**)&gpu_pt24, pt_size);
	hipMalloc((void**)&gpu_pt25, pt_size);
	hipMalloc((void**)&gpu_pt26, pt_size);
	hipMalloc((void**)&gpu_pt27, pt_size);
	hipMalloc((void**)&gpu_pt28, pt_size);
	hipMalloc((void**)&gpu_pt29, pt_size);
	hipMalloc((void**)&gpu_pt30, pt_size);
	hipMalloc((void**)&gpu_pt31, pt_size);

	hipMalloc((void**)&gpu_ha0, ha_size);
	hipMalloc((void**)&gpu_ha1, ha_size);
	hipMalloc((void**)&gpu_ha2, ha_size);
	hipMalloc((void**)&gpu_ha3, ha_size);
	hipMalloc((void**)&gpu_ha4, ha_size);
	hipMalloc((void**)&gpu_ha5, ha_size);
	hipMalloc((void**)&gpu_ha6, ha_size);
	hipMalloc((void**)&gpu_ha7, ha_size);
	hipMalloc((void**)&gpu_ha8, ha_size);
	hipMalloc((void**)&gpu_ha9, ha_size);
	hipMalloc((void**)&gpu_ha10, ha_size);
	hipMalloc((void**)&gpu_ha11, ha_size);
	hipMalloc((void**)&gpu_ha12, ha_size);
	hipMalloc((void**)&gpu_ha13, ha_size);
	hipMalloc((void**)&gpu_ha14, ha_size);
	hipMalloc((void**)&gpu_ha15, ha_size);
	hipMalloc((void**)&gpu_ha16, ha_size);
	hipMalloc((void**)&gpu_ha17, ha_size);
	hipMalloc((void**)&gpu_ha18, ha_size);
	hipMalloc((void**)&gpu_ha19, ha_size);
	hipMalloc((void**)&gpu_ha20, ha_size);
	hipMalloc((void**)&gpu_ha21, ha_size);
	hipMalloc((void**)&gpu_ha22, ha_size);
	hipMalloc((void**)&gpu_ha23, ha_size);
	hipMalloc((void**)&gpu_ha24, ha_size);
	hipMalloc((void**)&gpu_ha25, ha_size);
	hipMalloc((void**)&gpu_ha26, ha_size);
	hipMalloc((void**)&gpu_ha27, ha_size);
	hipMalloc((void**)&gpu_ha28, ha_size);
	hipMalloc((void**)&gpu_ha29, ha_size);
	hipMalloc((void**)&gpu_ha30, ha_size);
	hipMalloc((void**)&gpu_ha31, ha_size);
#endif

#if 1
	memset(pt0, 0, pt_size);
	memset(pt1, 1, pt_size);
	memset(pt2, 2, pt_size);
	memset(pt3, 3, pt_size);
	memset(pt4, 4, pt_size);
	memset(pt5, 5, pt_size);
	memset(pt6, 6, pt_size);
	memset(pt7, 7, pt_size);
	memset(pt8, 8, pt_size);
	memset(pt9, 9, pt_size);
	memset(pt10, 10, pt_size);
	memset(pt11, 11, pt_size);
	memset(pt12, 12, pt_size);
	memset(pt13, 13, pt_size);
	memset(pt14, 14, pt_size);
	memset(pt15, 15, pt_size);
	memset(pt16, 16, pt_size);
	memset(pt17, 17, pt_size);
	memset(pt18, 18, pt_size);
	memset(pt19, 19, pt_size);
	memset(pt20, 20, pt_size);
	memset(pt21, 21, pt_size);
	memset(pt22, 22, pt_size);
	memset(pt23, 23, pt_size);
	//memset(pt24, 24, pt_size);
	//memset(pt25, 25, pt_size);
	//memset(pt26, 26, pt_size);
	//memset(pt27, 27, pt_size);
	//memset(pt28, 28, pt_size);
	//memset(pt29, 29, pt_size);
	//memset(pt30, 30, pt_size);
	//memset(pt31, 0, pt_size);

	hipStreamCreate(&S[0]);
	hipStreamCreate(&S[1]);
	hipStreamCreate(&S[2]);
	hipStreamCreate(&S[3]);
	hipStreamCreate(&S[4]);
	hipStreamCreate(&S[5]);
	hipStreamCreate(&S[6]);
	hipStreamCreate(&S[7]);
	hipStreamCreate(&S[8]);
	hipStreamCreate(&S[9]);
	hipStreamCreate(&S[10]);
	hipStreamCreate(&S[11]);
	hipStreamCreate(&S[12]);
	hipStreamCreate(&S[13]);
	hipStreamCreate(&S[14]);
	hipStreamCreate(&S[15]);
	hipStreamCreate(&S[16]);
	hipStreamCreate(&S[17]);
	hipStreamCreate(&S[18]);
	hipStreamCreate(&S[19]);
	hipStreamCreate(&S[20]);
	hipStreamCreate(&S[21]);
	hipStreamCreate(&S[22]);
	hipStreamCreate(&S[23]);
	//hipStreamCreate(&S[24]);
	//hipStreamCreate(&S[25]);
	//hipStreamCreate(&S[26]);
	//hipStreamCreate(&S[27]);
	//hipStreamCreate(&S[28]);
	//hipStreamCreate(&S[29]);
	//hipStreamCreate(&S[30]);
	//hipStreamCreate(&S[31]);
#endif

#if 1
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		hipMemcpyAsync(gpu_pt0, pt0, pt_size, hipMemcpyHostToDevice, S[0]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[0] >> > (gpu_pt0, gpu_ha0);
		hipMemcpyAsync(ha0, gpu_ha0, ha_size, hipMemcpyDeviceToHost, S[0]);

		hipMemcpyAsync(gpu_pt1, pt1, pt_size, hipMemcpyHostToDevice, S[1]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[1] >> > (gpu_pt1, gpu_ha1);
		hipMemcpyAsync(ha1, gpu_ha1, ha_size, hipMemcpyDeviceToHost, S[1]);

		hipMemcpyAsync(gpu_pt2, pt2, pt_size, hipMemcpyHostToDevice, S[2]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[2] >> > (gpu_pt2, gpu_ha2);
		hipMemcpyAsync(ha2, gpu_ha2, ha_size, hipMemcpyDeviceToHost, S[2]);

		hipMemcpyAsync(gpu_pt3, pt3, pt_size, hipMemcpyHostToDevice, S[3]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[3] >> > (gpu_pt3, gpu_ha3);
		hipMemcpyAsync(ha3, gpu_ha3, ha_size, hipMemcpyDeviceToHost, S[3]);

		hipMemcpyAsync(gpu_pt4, pt4, pt_size, hipMemcpyHostToDevice, S[4]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[4] >> > (gpu_pt4, gpu_ha4);
		hipMemcpyAsync(ha4, gpu_ha4, ha_size, hipMemcpyDeviceToHost, S[4]);

		hipMemcpyAsync(gpu_pt5, pt5, pt_size, hipMemcpyHostToDevice, S[5]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[5] >> > (gpu_pt5, gpu_ha5);
		hipMemcpyAsync(ha5, gpu_ha5, ha_size, hipMemcpyDeviceToHost, S[5]);

		hipMemcpyAsync(gpu_pt6, pt6, pt_size, hipMemcpyHostToDevice, S[6]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[6] >> > (gpu_pt6, gpu_ha6);
		hipMemcpyAsync(ha6, gpu_ha6, ha_size, hipMemcpyDeviceToHost, S[6]);

		hipMemcpyAsync(gpu_pt7, pt7, pt_size, hipMemcpyHostToDevice, S[7]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[7] >> > (gpu_pt7, gpu_ha7);
		hipMemcpyAsync(ha7, gpu_ha7, ha_size, hipMemcpyDeviceToHost, S[7]);

		hipMemcpyAsync(gpu_pt8, pt8, pt_size, hipMemcpyHostToDevice, S[8]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[8] >> > (gpu_pt8, gpu_ha8);
		hipMemcpyAsync(ha8, gpu_ha8, ha_size, hipMemcpyDeviceToHost, S[8]);

		hipMemcpyAsync(gpu_pt9, pt9, pt_size, hipMemcpyHostToDevice, S[9]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[9] >> > (gpu_pt9, gpu_ha9);
		hipMemcpyAsync(ha9, gpu_ha9, ha_size, hipMemcpyDeviceToHost, S[9]);

		hipMemcpyAsync(gpu_pt10, pt10, pt_size, hipMemcpyHostToDevice, S[10]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[10] >> > (gpu_pt10, gpu_ha10);
		hipMemcpyAsync(ha10, gpu_ha10, ha_size, hipMemcpyDeviceToHost, S[10]);

		hipMemcpyAsync(gpu_pt11, pt11, pt_size, hipMemcpyHostToDevice, S[11]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[11] >> > (gpu_pt11, gpu_ha11);
		hipMemcpyAsync(ha11, gpu_ha11, ha_size, hipMemcpyDeviceToHost, S[11]);

		hipMemcpyAsync(gpu_pt12, pt12, pt_size, hipMemcpyHostToDevice, S[12]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[12] >> > (gpu_pt12, gpu_ha12);
		hipMemcpyAsync(ha12, gpu_ha12, ha_size, hipMemcpyDeviceToHost, S[12]);

		hipMemcpyAsync(gpu_pt13, pt13, pt_size, hipMemcpyHostToDevice, S[13]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[13] >> > (gpu_pt13, gpu_ha13);
		hipMemcpyAsync(ha13, gpu_ha13, ha_size, hipMemcpyDeviceToHost, S[13]);

		hipMemcpyAsync(gpu_pt14, pt14, pt_size, hipMemcpyHostToDevice, S[14]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[14] >> > (gpu_pt14, gpu_ha14);
		hipMemcpyAsync(ha14, gpu_ha14, ha_size, hipMemcpyDeviceToHost, S[14]);

		hipMemcpyAsync(gpu_pt15, pt15, pt_size, hipMemcpyHostToDevice, S[15]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[15] >> > (gpu_pt15, gpu_ha15);
		hipMemcpyAsync(ha15, gpu_ha15, ha_size, hipMemcpyDeviceToHost, S[15]);

		hipMemcpyAsync(gpu_pt16, pt16, pt_size, hipMemcpyHostToDevice, S[16]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[16] >> > (gpu_pt16, gpu_ha16);
		hipMemcpyAsync(ha16, gpu_ha16, ha_size, hipMemcpyDeviceToHost, S[16]);

		hipMemcpyAsync(gpu_pt17, pt17, pt_size, hipMemcpyHostToDevice, S[17]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[17] >> > (gpu_pt17, gpu_ha17);
		hipMemcpyAsync(ha17, gpu_ha17, ha_size, hipMemcpyDeviceToHost, S[17]);

		hipMemcpyAsync(gpu_pt18, pt18, pt_size, hipMemcpyHostToDevice, S[18]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[18] >> > (gpu_pt18, gpu_ha18);
		hipMemcpyAsync(ha18, gpu_ha18, ha_size, hipMemcpyDeviceToHost, S[18]);

		hipMemcpyAsync(gpu_pt19, pt19, pt_size, hipMemcpyHostToDevice, S[19]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[19] >> > (gpu_pt19, gpu_ha19);
		hipMemcpyAsync(ha19, gpu_ha19, ha_size, hipMemcpyDeviceToHost, S[19]);

		hipMemcpyAsync(gpu_pt20, pt20, pt_size, hipMemcpyHostToDevice, S[20]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[20] >> > (gpu_pt20, gpu_ha20);
		hipMemcpyAsync(ha20, gpu_ha20, ha_size, hipMemcpyDeviceToHost, S[20]);

		hipMemcpyAsync(gpu_pt21, pt21, pt_size, hipMemcpyHostToDevice, S[21]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[21] >> > (gpu_pt21, gpu_ha21);
		hipMemcpyAsync(ha21, gpu_ha21, ha_size, hipMemcpyDeviceToHost, S[21]);

		hipMemcpyAsync(gpu_pt22, pt22, pt_size, hipMemcpyHostToDevice, S[22]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[22] >> > (gpu_pt22, gpu_ha22);
		hipMemcpyAsync(ha22, gpu_ha22, ha_size, hipMemcpyDeviceToHost, S[22]);

		hipMemcpyAsync(gpu_pt23, pt23, pt_size, hipMemcpyHostToDevice, S[23]);
		cuda_lsh512 << <blocksize, threadsize, 0, S[23] >> > (gpu_pt23, gpu_ha23);
		hipMemcpyAsync(ha23, gpu_ha23, ha_size, hipMemcpyDeviceToHost, S[23]);

		//hipMemcpyAsync(gpu_pt24, pt24, pt_size, hipMemcpyHostToDevice, S[24]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[24] >> > (gpu_pt24, gpu_ha24);
		//hipMemcpyAsync(ha24, gpu_ha24, ha_size, hipMemcpyDeviceToHost, S[24]);

		//hipMemcpyAsync(gpu_pt25, pt25, pt_size, hipMemcpyHostToDevice, S[25]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[25] >> > (gpu_pt25, gpu_ha25);
		//hipMemcpyAsync(ha25, gpu_ha25, ha_size, hipMemcpyDeviceToHost, S[25]);

		//hipMemcpyAsync(gpu_pt26, pt26, pt_size, hipMemcpyHostToDevice, S[26]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[26] >> > (gpu_pt26, gpu_ha26);
		//hipMemcpyAsync(ha26, gpu_ha26, ha_size, hipMemcpyDeviceToHost, S[26]);

		//hipMemcpyAsync(gpu_pt27, pt27, pt_size, hipMemcpyHostToDevice, S[27]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[27] >> > (gpu_pt27, gpu_ha27);
		//hipMemcpyAsync(ha27, gpu_ha27, ha_size, hipMemcpyDeviceToHost, S[27]);

		//hipMemcpyAsync(gpu_pt28, pt28, pt_size, hipMemcpyHostToDevice, S[28]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[28] >> > (gpu_pt28, gpu_ha28);
		//hipMemcpyAsync(ha28, gpu_ha28, ha_size, hipMemcpyDeviceToHost, S[28]);

		//hipMemcpyAsync(gpu_pt29, pt29, pt_size, hipMemcpyHostToDevice, S[29]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[29] >> > (gpu_pt29, gpu_ha29);
		//hipMemcpyAsync(ha29, gpu_ha29, ha_size, hipMemcpyDeviceToHost, S[29]);

		//hipMemcpyAsync(gpu_pt30, pt30, pt_size, hipMemcpyHostToDevice, S[30]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[30] >> > (gpu_pt30, gpu_ha30);
		//hipMemcpyAsync(ha30, gpu_ha30, ha_size, hipMemcpyDeviceToHost, S[30]);

		//hipMemcpyAsync(gpu_pt31, pt31, pt_size, hipMemcpyHostToDevice, S[31]);
		//cuda_lsh512 << <blocksize, threadsize, 0, S[31] >> > (gpu_pt31, gpu_ha31);
		//hipMemcpyAsync(ha31, gpu_ha31, ha_size, hipMemcpyDeviceToHost, S[31]);

		hipStreamSynchronize(S[0]);
		hipStreamSynchronize(S[1]);
		hipStreamSynchronize(S[2]);
		hipStreamSynchronize(S[3]);
		hipStreamSynchronize(S[4]);
		hipStreamSynchronize(S[5]);
		hipStreamSynchronize(S[6]);
		hipStreamSynchronize(S[7]);
		hipStreamSynchronize(S[8]);
		hipStreamSynchronize(S[9]);
		hipStreamSynchronize(S[10]);
		hipStreamSynchronize(S[11]);
		hipStreamSynchronize(S[12]);
		hipStreamSynchronize(S[13]);
		hipStreamSynchronize(S[14]);
		hipStreamSynchronize(S[15]);
		hipStreamSynchronize(S[16]);
		hipStreamSynchronize(S[17]);
		hipStreamSynchronize(S[18]);
		hipStreamSynchronize(S[19]);
		hipStreamSynchronize(S[20]);
		hipStreamSynchronize(S[21]);
		hipStreamSynchronize(S[22]);
		hipStreamSynchronize(S[23]);
		//hipStreamSynchronize(S[24]);
		//hipStreamSynchronize(S[25]);
		//hipStreamSynchronize(S[26]);
		//hipStreamSynchronize(S[27]);
		//hipStreamSynchronize(S[28]);
		//hipStreamSynchronize(S[29]);
		//hipStreamSynchronize(S[30]);
		//hipStreamSynchronize(S[31]);
	}
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);

	elapsed_time_ms = elapsed_time_ms / 100;
	elapsed_time_ms /= 24;
	elapsed_time_ms = 1000 / elapsed_time_ms;
	elapsed_time_ms = blocksize * threadsize * elapsed_time_ms;
	elapsed_time_ms /= 1000;
	elapsed_time_ms /= 1000;
	printf("operation time (%d, %d) = %4.2fMH/s\n", blocksize, threadsize, elapsed_time_ms);
	printf("%02X \n", ha31[0]);
	hipStreamDestroy(S[0]);
	hipStreamDestroy(S[1]);
	hipStreamDestroy(S[2]);
	hipStreamDestroy(S[3]);
	hipStreamDestroy(S[4]);
	hipStreamDestroy(S[5]);
	hipStreamDestroy(S[6]);
	hipStreamDestroy(S[7]);
	hipStreamDestroy(S[8]);
	hipStreamDestroy(S[9]);
	hipStreamDestroy(S[10]);
	hipStreamDestroy(S[11]);
	hipStreamDestroy(S[12]);
	hipStreamDestroy(S[13]);
	hipStreamDestroy(S[14]);
	hipStreamDestroy(S[15]);
	hipStreamDestroy(S[16]);
	hipStreamDestroy(S[17]);
	hipStreamDestroy(S[18]);
	hipStreamDestroy(S[19]);
	hipStreamDestroy(S[20]);
	hipStreamDestroy(S[21]);
	hipStreamDestroy(S[22]);
	hipStreamDestroy(S[23]);
	//hipStreamDestroy(S[24]);
	//hipStreamDestroy(S[25]);
	//hipStreamDestroy(S[26]);
	//hipStreamDestroy(S[27]);
	//hipStreamDestroy(S[28]);
	//hipStreamDestroy(S[29]);
	//hipStreamDestroy(S[30]);
	//hipStreamDestroy(S[31]);
#endif
	hipFree(gpu_pt0);
	hipFree(gpu_pt1);
	hipFree(gpu_pt2);
	hipFree(gpu_pt3);
	hipFree(gpu_pt4);
	hipFree(gpu_pt5);
	hipFree(gpu_pt6);
	hipFree(gpu_pt7);
	hipFree(gpu_pt8);
	hipFree(gpu_pt9);
	hipFree(gpu_pt10);
	hipFree(gpu_pt11);
	hipFree(gpu_pt12);
	hipFree(gpu_pt13);
	hipFree(gpu_pt14);
	hipFree(gpu_pt15);
	hipFree(gpu_pt16);
	hipFree(gpu_pt17);
	hipFree(gpu_pt18);
	hipFree(gpu_pt19);
	hipFree(gpu_pt20);
	hipFree(gpu_pt21);
	hipFree(gpu_pt22);
	hipFree(gpu_pt23);
	//hipFree(gpu_pt24);
	//hipFree(gpu_pt25);
	//hipFree(gpu_pt26);
	//hipFree(gpu_pt27);
	//hipFree(gpu_pt28);
	//hipFree(gpu_pt29);
	//hipFree(gpu_pt30);
	//hipFree(gpu_pt31);

	hipFree(gpu_ha0);
	hipFree(gpu_ha1);
	hipFree(gpu_ha2);
	hipFree(gpu_ha3);
	hipFree(gpu_ha4);
	hipFree(gpu_ha5);
	hipFree(gpu_ha6);
	hipFree(gpu_ha7);
	hipFree(gpu_ha8);
	hipFree(gpu_ha9);
	hipFree(gpu_ha10);
	hipFree(gpu_ha11);
	hipFree(gpu_ha12);
	hipFree(gpu_ha13);
	hipFree(gpu_ha14);
	hipFree(gpu_ha15);
	hipFree(gpu_ha16);
	hipFree(gpu_ha17);
	hipFree(gpu_ha18);
	hipFree(gpu_ha19);
	hipFree(gpu_ha20);
	hipFree(gpu_ha21);
	hipFree(gpu_ha22);
	hipFree(gpu_ha23);
	//hipFree(gpu_ha24);
	//hipFree(gpu_ha25);
	//hipFree(gpu_ha26);
	//hipFree(gpu_ha27);
	//hipFree(gpu_ha28);
	//hipFree(gpu_ha29);
	//hipFree(gpu_ha30);
	//hipFree(gpu_ha31);

	hipHostFree(pt0);
	hipHostFree(pt1);
	hipHostFree(pt2);
	hipHostFree(pt3);
	hipHostFree(pt4);
	hipHostFree(pt5);
	hipHostFree(pt6);
	hipHostFree(pt7);
	hipHostFree(pt8);
	hipHostFree(pt9);
	hipHostFree(pt10);
	hipHostFree(pt11);
	hipHostFree(pt12);
	hipHostFree(pt13);
	hipHostFree(pt14);
	hipHostFree(pt15);
	hipHostFree(pt16);
	hipHostFree(pt17);
	hipHostFree(pt18);
	hipHostFree(pt19);
	hipHostFree(pt20);
	hipHostFree(pt21);
	hipHostFree(pt22);
	hipHostFree(pt23);
	//hipHostFree(pt24);
	//hipHostFree(pt25);
	//hipHostFree(pt26);
	//hipHostFree(pt27);
	//hipHostFree(pt28);
	//hipHostFree(pt29);
	//hipHostFree(pt30);
	//hipHostFree(pt31);

	hipHostFree(ha0);
	hipHostFree(ha1);
	hipHostFree(ha2);
	hipHostFree(ha3);
	hipHostFree(ha4);
	hipHostFree(ha5);
	hipHostFree(ha6);
	hipHostFree(ha7);
	hipHostFree(ha8);
	hipHostFree(ha9);
	hipHostFree(ha10);
	hipHostFree(ha11);
	hipHostFree(ha12);
	hipHostFree(ha13);
	hipHostFree(ha14);
	hipHostFree(ha15);
	hipHostFree(ha16);
	hipHostFree(ha17);
	hipHostFree(ha18);
	hipHostFree(ha19);
	hipHostFree(ha20);
	hipHostFree(ha21);
	hipHostFree(ha22);
	hipHostFree(ha23);
	//hipHostFree(ha24);
	//hipHostFree(ha25);
	//hipHostFree(ha26);
	//hipHostFree(ha27);
	//hipHostFree(ha28);
	//hipHostFree(ha29);
	//hipHostFree(ha30);
	//hipHostFree(ha31);
}

int main() {
	printf("main function start\n");
	uint32_t block_value[17] = { 1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536 };
	uint32_t thread_value[15] = { 1, 2, 4, 8, 16, 32, 64, 128, 256, 384, 512, 640, 768, 896, 1024 };
	cudastream_test(4096, 128);

	/*for (int j = 6; j < 16; j++) {
		cudastream_test(block_value[j], 128);
	}*/
	return 0;
}